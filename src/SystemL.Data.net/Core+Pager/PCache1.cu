#include "hip/hip_runtime.h"
﻿// pcache1.c
#include "Core+Pager.cu.h"
#include <new.h>

namespace Core
{
#pragma region Struct

	typedef struct PgHdr1 PgHdr1;

	struct PGroup 
	{
		MutexEx Mutex;					// MUTEX_STATIC_LRU or NULL
		uint MaxPages;					// Sum of nMax for purgeable caches
		uint MinPages;					// Sum of nMin for purgeable caches
		uint MaxPinned;					// nMaxpage + 10 - nMinPage
		uint CurrentPages;				// Number of purgeable pages allocated
		PgHdr1 *LruHead, *LruTail;		// LRU list of unpinned pages
	};

	class PCache1 : IPCache
	{
	public:
		// Cache configuration parameters. Page size (szPage) and the purgeable flag (bPurgeable) are set when the cache is created. nMax may be 
		// modified at any time by a call to the pcache1Cachesize() method. The PGroup mutex must be held when accessing nMax.
		PGroup *Group;			// PGroup this cache belongs to
		int SizePage;           // Size of allocated pages in bytes
		int SizeExtra;          // Size of extra space in bytes
		bool Purgeable;			// True if cache is purgeable
		uint Min;				// Minimum number of pages reserved
		uint Max;				// Configured "cache_size" value
		uint N90pct;			// nMax*9/10
		uint MaxID;				// Largest key seen since xTruncate()
		// Hash table of all pages. The following variables may only be accessed when the accessor is holding the PGroup mutex.
		uint Recyclables;       // Number of pages in the LRU list
		uint Pages;             // Total number of pages in apHash
		array_t<PgHdr1 *>Hash;	// Hash table for fast lookup by key
	public:
		//static void *PageAlloc(int size);
		//static void PageFree(void *p);
		//bool UnderMemoryPressure();
		//
		__device__ RC Init();
		__device__ void Shutdown();
		__device__ IPCache *Create(int sizePage, int sizeExtra, bool purgeable);
		__device__ void Cachesize(uint max);
		__device__ void Shrink();
		__device__ int get_Pages();
		__device__ ICachePage *Fetch(Pid id, bool createFlag);
		__device__ void Unpin(ICachePage *pg, bool reuseUnlikely);
		__device__ void Rekey(ICachePage *pg, Pid old, Pid new_);
		__device__ void Truncate(Pid limit);
		__device__ void Destroy(IPCache *p);
	};

	struct PgHdr1
	{
		ICachePage Page;
		Pid ID;					// Key value (page number)
		PgHdr1 *Next;			// Next in hash table chain
		PCache1 *Cache;			// Cache that currently owns this page
		PgHdr1 *LruNext;		// Next in LRU list of unpinned pages
		PgHdr1 *LruPrev;		// Previous in LRU list of unpinned pages
	};

	struct PgFreeslot
	{
		PgFreeslot *Next;		// Next free slot
	};

	struct PCacheGlobal
	{
		PGroup Group;			// The global PGroup for mode (2)
		// Variables related to SQLITE_CONFIG_PAGECACHE settings.  The szSlot, nSlot, pStart, pEnd, nReserve, and isInit values are all
		// fixed at sqlite3_initialize() time and do not require mutex protection. The nFreeSlot and pFree values do require mutex protection.
		bool IsInit;			// True if initialized
		int SizeSlot;			// Size of each free slot
		int Slots;				// The number of pcache slots
		int Reserves;			// Try to keep nFreeSlot above this
		void *Start, *End;		// Bounds of pagecache malloc range
		// Above requires no mutex.  Use mutex below for variable that follow.
		MutexEx Mutex;			// Mutex for accessing the following:
		PgFreeslot *Free;		// Free page blocks
		int FreeSlots;			// Number of unused pcache slots
		// The following value requires a mutex to change.  We skip the mutex on reading because (1) most platforms read a 32-bit integer atomically and
		// (2) even if an incorrect value is read, no great harm is done since this is really just an optimization.
		bool UnderPressure;		// True if low on PAGECACHE memory
	};

#pragma endregion

	__device__ static _WSD PCacheGlobal g_pcache1;
	#define _pcache1 _GLOBAL(PCacheGlobal, g_pcache1)

#pragma region Page Allocation

	__device__ void BufferSetup(void *buffer, int size, int n)
	{
		if (_pcache1.IsInit)
		{
			size = _ROUNDDOWN8(size);
			_pcache1.SizeSlot = size;
			_pcache1.Slots = _pcache1.FreeSlots = n;
			_pcache1.Reserves = (n > 90 ? 10 : (n / 10 + 1));
			_pcache1.Start = buffer;
			_pcache1.Free = nullptr;
			_pcache1.UnderPressure = false;
			while (n--)
			{
				PgFreeslot *p = (PgFreeslot *)buffer;
				p->Next = _pcache1.Free;
				_pcache1.Free = p;
				buffer = (void *)&((char *)buffer)[size];
			}
			_pcache1.End = buffer;
		}
	}

	__device__ void *Alloc(int bytes)
	{
		_assert(MutexEx::NotHeld(_pcache1.Group.Mutex));
		StatusEx::StatusSet(StatusEx::STATUS_PAGECACHE_SIZE, bytes);
		void *p = nullptr;
		if (bytes <= _pcache1.SizeSlot)
		{
			MutexEx::Enter(_pcache1.Mutex);
			p = (PgHdr1 *)_pcache1.Free;
			if (p)
			{
				_pcache1.Free = _pcache1.Free->Next;
				_pcache1.FreeSlots--;
				_pcache1.UnderPressure = (_pcache1.FreeSlots < _pcache1.Reserves);
				_assert(_pcache1.FreeSlots >= 0);
				StatusEx::StatusAdd(StatusEx::STATUS_PAGECACHE_USED, 1);
			}
			MutexEx::Leave(_pcache1.Mutex);
		}
		if (!p)
		{
			// Memory is not available in the SQLITE_CONFIG_PAGECACHE pool.  Get it from sqlite3Malloc instead.
			p = _alloc(bytes);
#ifndef DISABLE_PAGECACHE_OVERFLOW_STATS
			if (p)
			{
				int size = _allocsize(p);
				MutexEx::Enter(_pcache1.Mutex);
				StatusEx::StatusAdd(StatusEx::STATUS_PAGECACHE_OVERFLOW, size);
				MutexEx::Leave(_pcache1.Mutex);
			}
#endif
			_memdbg_settype(p, MEMTYPE_PCACHE);
		}
		return p;
	}

	__device__ int Free(void *p)
	{
		int freed = 0;
		if (p == nullptr)
			return 0;
		if (p >= _pcache1.Start && p < _pcache1.End)
		{
			MutexEx::Enter(_pcache1.Mutex);
			StatusEx::StatusAdd(StatusEx::STATUS_PAGECACHE_USED, -1);
			PgFreeslot *slot = (PgFreeslot *)p;
			slot->Next = _pcache1.Free;
			_pcache1.Free = slot;
			_pcache1.FreeSlots++;
			_pcache1.UnderPressure = (_pcache1.FreeSlots < _pcache1.Reserves);
			_assert(_pcache1.FreeSlots <= _pcache1.Slots);
			MutexEx::Leave(_pcache1.Mutex);
		}
		else
		{
			_assert(_memdbg_hastype(p, MEMTYPE_PCACHE));
			_memdbg_settype(p, MEMTYPE_HEAP);
			freed = _allocsize(p);
#ifndef DISABLE_PAGECACHE_OVERFLOW_STATS
			MutexEx::Enter(_pcache1.Mutex);
			StatusEx::StatusAdd(StatusEx::STATUS_PAGECACHE_OVERFLOW, -freed);
			MutexEx::Leave(_pcache1.Mutex);
#endif
			_free(p);
		}
		return freed;
	}

#ifdef ENABLE_MEMORY_MANAGEMENT
	__device__ static int MemSize(void *p)
	{
		if (p >= _pcache1.Start && p < _pcache1.End)
			return _pcache1.SizeSlot;
		_assert(_memdbg_hastype(p, MEMTYPE_PCACHE));
		_memdbg_settype(p, MEMTYPE_HEAP);
		int size = _allocsize(p);
		_memdbg_settype(p, MEMTYPE_PCACHE);
		return size;
	}
#endif

	__device__ static PgHdr1 *AllocPage(PCache1 *cache)
	{
		// The group mutex must be released before pcache1Alloc() is called. This is because it may call sqlite3_release_memory(), which assumes that this mutex is not held.
		_assert(MutexEx::Held(cache->Group->Mutex));
		MutexEx::Leave(cache->Group->Mutex);
		PgHdr1 *p = nullptr;
		void *pg;
#ifdef PCACHE_SEPARATE_HEADER
		pg = Alloc(cache->SizePage);
		p = (PgHdr1 *)_alloc(sizeof(PgHdr1) + cache->SizeExtra);
		if (!pg || !p)
		{
			Free(pg);
			_free(p);
			pg = nullptr;
		}
#else
		pg = Alloc(sizeof(PgHdr1) + cache->SizePage + cache->SizeExtra);
		p = (PgHdr1 *)&((uint8 *)pg)[cache->SizePage];
#endif
		MutexEx::Enter(cache->Group->Mutex);
		if (pg)
		{
			p->Page.Buffer = pg;
			p->Page.Extra = &p[1];
			if (cache->Purgeable)
				cache->Group->CurrentPages++;
			return p;
		}
		return nullptr;
	}

	__device__ static void FreePage(PgHdr1 *p)
	{
		if (_ALWAYS(p))
		{
			PCache1 *cache = p->Cache;
			_assert(MutexEx::Held(p->Cache->Group->Mutex));
			Free(p->Page.Buffer);
#ifdef PCACHE_SEPARATE_HEADER
			_free(p);
#endif
			if (cache->Purgeable)
				cache->Group->CurrentPages--;
		}
	}

	__device__ static bool UnderMemoryPressure(PCache1 *cache)
	{
		return (_pcache1.Slots && (cache->SizePage + cache->SizeExtra) <= _pcache1.SizeSlot ? _pcache1.UnderPressure : _heapnearlyfull());
	}

#pragma endregion

#pragma region General

	__device__ static int ResizeHash(PCache1 *p)
	{
		_assert(MutexEx::Held(p->Group->Mutex));
		uint newLength = p->Hash.length * 2;
		if (newLength < 256)
			newLength = 256;
		MutexEx::Leave(p->Group->Mutex);
		if (p->Hash.length) _benignalloc_begin();
		PgHdr1 **newHash = (PgHdr1 **)_alloc2(sizeof(PgHdr1 *) * newLength, true);
		if (p->Hash.length) _benignalloc_end();
		MutexEx::Enter(p->Group->Mutex);
		if (newHash)
		{
			for (uint i = 0; i < (uint)p->Hash.length; i++)
			{
				PgHdr1 *page;
				PgHdr1 *next = p->Hash[i];
				while ((page = next) != 0)
				{
					uint h = (page->ID % newLength);
					next = page->Next;
					page->Next = newHash[h];
					newHash[h] = page;
				}
			}
			_free(p->Hash);
			p->Hash = newHash;
			p->Hash.length = newLength;
		}
		return (p->Hash ? RC_OK : RC_NOMEM);
	}

	__device__ static void PinPage(PgHdr1 *page)
	{
		if (page == nullptr)
			return;
		PCache1 *cache = page->Cache;
		PGroup *group = cache->Group;
		_assert(MutexEx::Held(group->Mutex));
		if (page->LruNext || page == group->LruTail)
		{
			if (page->LruPrev)
				page->LruPrev->LruNext = page->LruNext;
			if (page->LruNext)
				page->LruNext->LruPrev = page->LruPrev;
			if (group->LruHead == page)
				group->LruHead = page->LruNext;
			if (group->LruTail == page)
				group->LruTail = page->LruPrev;
			page->LruNext = 0;
			page->LruPrev = 0;
			page->Cache->Recyclables--;
		}
	}

	__device__ static void RemoveFromHash(PgHdr1 *page)
	{
		PCache1 *cache = page->Cache;
		_assert(MutexEx::Held(cache->Group->Mutex));
		uint h = (page->ID % cache->Hash.length);
		PgHdr1 **pp;
		for (pp = &cache->Hash[h]; (*pp) != page; pp = &(*pp)->Next);
		*pp = (*pp)->Next;
		cache->Pages--;
	}

	__device__ static void EnforceMaxPage(PGroup *group)
	{
		_assert(MutexEx::Held(group->Mutex));
		while (group->CurrentPages > group->MaxPages && group->LruTail)
		{
			PgHdr1 *p = group->LruTail;
			_assert(p->Cache->Group == group);
			PinPage(p);
			RemoveFromHash(p);
			FreePage(p);
		}
	}

	__device__ static void TruncateUnsafe(PCache1 *p, Pid limit)
	{
		ASSERTONLY(uint pages = 0;)
			_assert(MutexEx::Held(p->Group->Mutex));
		for (uint h = 0; h < (uint)p->Hash.length; h++)
		{
			PgHdr1 **pp = &p->Hash[h]; 
			PgHdr1 *page;
			while ((page = *pp) != 0)
			{
				if (page->ID >= limit)
				{
					p->Pages--;
					*pp = page->Next;
					PinPage(page);
					FreePage(page);
				}
				else
				{
					pp = &page->Next;
					ASSERTONLY(pages++;)
				}
			}
		}
		_assert(p->Pages == pages);
	}

#pragma endregion

#pragma region Interface

	__device__ IPCache *new_PCache1() { PCache1 *cache = (PCache1 *)_alloc2(sizeof(PCache1), true); return (IPCache *)(new (cache) PCache1()); }

	__device__ RC PCache1::Init()
	{
		_assert(!_pcache1.IsInit);
		_memset(&_pcache1, 0, sizeof(_pcache1));
		if (SysEx_GlobalStatics.CoreMutex)
		{
			_pcache1.Group.Mutex = MutexEx::Alloc(MutexEx::MUTEX_STATIC_LRU);
			_pcache1.Mutex = MutexEx::Alloc(MutexEx::MUTEX_STATIC_PMEM);
		}
		_pcache1.Group.MaxPinned = 10;
		_pcache1.IsInit = true;
		return RC_OK;
	}

	__device__ void PCache1::Shutdown()
	{
		_assert(_pcache1.IsInit);
		_memset(&_pcache1, 0, sizeof(_pcache1));
	}

	__device__ IPCache *PCache1::Create(int sizePage, int sizeExtra, bool purgeable)
	{
		// The seperateCache variable is true if each PCache has its own private PGroup.  In other words, separateCache is true for mode (1) where no
		// mutexing is required.
		// *  Always use a unified cache (mode-2) if ENABLE_MEMORY_MANAGEMENT
		// *  Always use a unified cache in single-threaded applications
		// *  Otherwise (if multi-threaded and ENABLE_MEMORY_MANAGEMENT is off) use separate caches (mode-1)
#if defined(ENABLE_MEMORY_MANAGEMENT) || THREADSAFE == 0
		const bool separateCache = false;
#else
		bool separateCache = SysEx_GlobalStatics.CoreMutex;
#endif
		_assert((sizePage & (sizePage - 1)) == 0 && sizePage >= 512 && sizePage <= 65536);
		_assert(sizeExtra < 300);
		int size = sizeof(PCache1) + sizeof(PGroup) * (int)separateCache;
		PCache1 *cache = (PCache1 *)_alloc2(size, true);
		cache = new (cache) PCache1();
		if (cache)
		{
			PGroup *group;
			if (separateCache)
			{
				group = (PGroup*)&cache[1];
				group->MaxPinned = 10;
			}
			else
				group = &_pcache1.Group;
			cache->Group = group;
			cache->SizePage = sizePage;
			cache->SizeExtra = sizeExtra;
			cache->Purgeable = purgeable;
			if (purgeable)
			{
				cache->Min = 10;
				MutexEx::Enter(group->Mutex);
				group->MinPages += cache->Min;
				group->MaxPinned = group->MaxPages + 10 - group->MinPages;
				MutexEx::Leave(group->Mutex);
			}
		}
		return (IPCache *)cache;
	}

	__device__ void PCache1::Cachesize(uint max)
	{
		if (Purgeable)
		{
			PGroup *group = Group;
			MutexEx::Enter(group->Mutex);
			group->MaxPages += (max - Max);
			group->MaxPinned = group->MaxPages + 10 - group->MinPages;
			Max = max;
			N90pct = Max * 9 / 10;
			EnforceMaxPage(group);
			MutexEx::Leave(group->Mutex);
		}
	}

	__device__ void PCache1::Shrink()
	{
		if (Purgeable)
		{
			PGroup *group = Group;
			MutexEx::Enter(group->Mutex);
			uint savedMaxPages = group->MaxPages;
			group->MaxPages = 0;
			EnforceMaxPage(group);
			group->MaxPages = savedMaxPages;
			MutexEx::Leave(group->Mutex);
		}
	}

	__device__ int PCache1::get_Pages()
	{
		MutexEx::Enter(Group->Mutex);
		int pages = Pages;
		MutexEx::Leave(Group->Mutex);
		return pages;
	}

	__device__ ICachePage *PCache1::Fetch(Pid id, bool createFlag)
	{
		_assert(Purgeable || !createFlag);
		_assert(Purgeable || Min == 0);
		_assert(!Purgeable || Min == 10);
		PGroup *group;
		MutexEx::Enter((group = Group)->Mutex);

		// Step 1: Search the hash table for an existing entry.
		PgHdr1 *page = nullptr;
		if (Hash.length > 0)
		{
			uint h = (id % Hash.length);
			for (page = Hash[h]; page && page->ID != id; page = page->Next) ;
		}

		// Step 2: Abort if no existing page is found and createFlag is 0
		uint pinned;
		if (page || !createFlag)
		{
			PinPage(page);
			goto fetch_out;
		}

		// The pGroup local variable will normally be initialized by the pcache1EnterMutex() macro above.  But if SQLITE_MUTEX_OMIT is defined,
		// then pcache1EnterMutex() is a no-op, so we have to initialize the local variable here.  Delaying the initialization of pGroup is an
		// optimization:  The common case is to exit the module before reaching this point.
#ifdef MUTEX_OMIT
		group = cache->Group;
#endif

		// Step 3: Abort if createFlag is 1 but the cache is nearly full
		_assert(Pages >= Recyclables);
		pinned = Pages - Recyclables;	
		_assert(group->MaxPinned == group->MaxPages + 10 - group->MinPages);
		_assert(N90pct == Max * 9 / 10);
		if (createFlag && (pinned >= group->MaxPinned || pinned >= N90pct || UnderMemoryPressure(this)))
			goto fetch_out;
		if (Pages >= (Pid)Hash.length && ResizeHash(this))
			goto fetch_out;

		// Step 4. Try to recycle a page.
		if (Purgeable && group->LruTail && ((Pages + 1 >= Max) || group->CurrentPages >= group->MaxPages || UnderMemoryPressure(this)))
		{
			page = group->LruTail;
			RemoveFromHash(page);
			PinPage(page);
			PCache1 *other = page->Cache;

			// We want to verify that szPage and szExtra are the same for pOther and pCache.  Assert that we can verify this by comparing sums.
			_assert((SizePage & (SizePage - 1)) == 0 && SizePage >= 512);
			_assert(SizeExtra < 512);
			_assert((other->SizePage & (other->SizePage - 1)) == 0 && other->SizePage >= 512);
			_assert(other->SizeExtra < 512);

			if (other->SizePage + other->SizeExtra != SizePage + SizeExtra)
			{
				FreePage(page);
				page = nullptr;
			}
			else
				group->CurrentPages -= (other->Purgeable - Purgeable);
		}

		// Step 5. If a usable page buffer has still not been found, attempt to allocate a new one. 
		if (!page)
		{
			if (createFlag) _benignalloc_begin();
			page = AllocPage(this);
			if (createFlag) _benignalloc_end();
		}
		if (page)
		{
			uint h = (id % Hash.length);
			Pages++;
			page->ID = id;
			page->Next = Hash[h];
			page->Cache = this;
			page->LruPrev = nullptr;
			page->LruNext = nullptr;
			*(void **)page->Page.Extra = nullptr;
			Hash[h] = page;
		}

fetch_out:
		if (page && id > MaxID)
			MaxID = id;
		MutexEx::Leave(group->Mutex);
		return &page->Page;
	}

	__device__ void PCache1::Unpin(ICachePage *pg, bool reuseUnlikely)
	{
		PgHdr1 *page = (PgHdr1 *)pg;
		PGroup *group = Group;
		_assert(page->Cache == this);
		MutexEx::Enter(group->Mutex);
		// It is an error to call this function if the page is already part of the PGroup LRU list.
		_assert(page->LruPrev == nullptr && page->LruNext == nullptr);
		_assert(group->LruHead != page && group->LruTail != page);
		if (reuseUnlikely || group->CurrentPages > group->MaxPages)
		{
			RemoveFromHash(page);
			FreePage(page);
		}
		else
		{
			// Add the page to the PGroup LRU list.
			if (group->LruHead)
			{
				group->LruHead->LruPrev = page;
				page->LruNext = group->LruHead;
				group->LruHead = page;
			}
			else
			{
				group->LruTail = page;
				group->LruHead = page;
			}
			Recyclables++;
		}
		MutexEx::Leave(Group->Mutex);
	}

	__device__ void PCache1::Rekey(ICachePage *pg, Pid old, Pid new_)
	{
		PgHdr1 *page = (PgHdr1 *)pg;
		_assert(page->ID == old);
		_assert(page->Cache == this);
		MutexEx::Enter(Group->Mutex);
		uint h = (old % Hash.length);
		PgHdr1 **pp = &Hash[h];
		while ((*pp) != page)
			pp = &(*pp)->Next;
		*pp = page->Next;
		h = (new_ % Hash.length);
		page->ID = new_;
		page->Next = Hash[h];
		Hash[h] = page;
		if (new_ > MaxID)
			MaxID = new_;
		MutexEx::Leave(Group->Mutex);
	}

	__device__ void PCache1::Truncate(Pid limit)
	{
		MutexEx::Enter(Group->Mutex);
		if (limit <= MaxID)
		{
			TruncateUnsafe(this, limit);
			MaxID = limit - 1;
		}
		MutexEx::Leave(Group->Mutex);
	}

	__device__ void PCache1::Destroy(IPCache *p)
	{
		PCache1 *cache = (PCache1 *)p;
		PGroup *group = cache->Group;
		_assert(cache->Purgeable || (cache->Max == 0 && cache->Min == 0));
		MutexEx::Enter(group->Mutex);
		TruncateUnsafe(cache, 0);
		_assert(group->MaxPages >= cache->Max);
		group->MaxPages -= cache->Max;
		_assert(group->MinPages >= cache->Min);
		group->MinPages -= cache->Min;
		group->MaxPinned = group->MaxPages + 10 - group->MinPages;
		EnforceMaxPage(group);
		MutexEx::Leave(group->Mutex);
		_free(cache->Hash);
		_free(cache);
	}

#ifdef ENABLE_MEMORY_MANAGEMENT
	__device__ int PCache::ReleaseMemory(int required)
	{
		_assert(MutexEx::NotHeld(_pcache1.Group.Mutex));
		_assert(MutexEx::NotHeld(_pcache1.Mutex));
		int free = 0;
		if (_pcache1.Start == nullptr)
		{
			PgHdr1 *p;
			MutexEx::Enter(_pcache1.Group.Mutex);
			while ((required < 0 || free < required) && ((p = _pcache1.Group.LruTail) != nullptr))
			{
				free += MemSize(p->Page.Buffer);
#ifdef PCACHE_SEPARATE_HEADER
				free += MemSize(p);
#endif
				PinPage(p);
				RemoveFromHash(p);
				FreePage(p);
			}
			MutexEx::Leave(_pcache1.Group.Mutex);
		}
		return free;
	}
#endif

#pragma endregion

#pragma	region Tests
#ifdef TEST

	__device__ void PCache1_testStats(uint *current, uint *max, uint *min, uint *recyclables)
	{
		uint recyclables2 = 0;
		for (PgHdr1 *p = _pcache1.Group.LruHead; p; p = p->LruNext)
			recyclables2++;
		*current = _pcache1.Group.CurrentPages;
		*max = _pcache1.Group.MaxPages;
		*min = _pcache1.Group.MinPages;
		*recyclables = recyclables2;
	}

#endif
#pragma endregion
}
