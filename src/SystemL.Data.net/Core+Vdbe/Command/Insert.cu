#include "hip/hip_runtime.h"
// insert.c
#include "..\Core+Vdbe.cu.h"
#include "..\VdbeInt.h"

namespace Core { namespace Command
{
	void sqlite3OpenTable(Parse *p, int iCur, int iDb, Table *pTab, int opcode)
	{
		Vdbe *v;
		assert( !IsVirtual(pTab) );
		v = sqlite3GetVdbe(p);
		assert( opcode==OP_OpenWrite || opcode==OP_OpenRead );
		sqlite3TableLock(p, iDb, pTab->tnum, (opcode==OP_OpenWrite)?1:0, pTab->zName);
		sqlite3VdbeAddOp3(v, opcode, iCur, pTab->tnum, iDb);
		sqlite3VdbeChangeP4(v, -1, SQLITE_INT_TO_PTR(pTab->nCol), P4_INT32);
		VdbeComment((v, "%s", pTab->zName));
	}

	const char *sqlite3IndexAffinityStr(Vdbe *v, Index *pIdx)
	{
		if( !pIdx->zColAff ){
			/* The first time a column affinity string for a particular index is
			** required, it is allocated and populated here. It is then stored as
			** a member of the Index structure for subsequent use.
			**
			** The column affinity string will eventually be deleted by
			** sqliteDeleteIndex() when the Index structure itself is cleaned
			** up.
			*/
			int n;
			Table *pTab = pIdx->pTable;
			sqlite3 *db = sqlite3VdbeDb(v);
			pIdx->zColAff = (char *)sqlite3DbMallocRaw(0, pIdx->nColumn+2);
			if( !pIdx->zColAff ){
				db->mallocFailed = 1;
				return 0;
			}
			for(n=0; n<pIdx->nColumn; n++){
				pIdx->zColAff[n] = pTab->aCol[pIdx->aiColumn[n]].affinity;
			}
			pIdx->zColAff[n++] = SQLITE_AFF_INTEGER;
			pIdx->zColAff[n] = 0;
		}

		return pIdx->zColAff;
	}

	void sqlite3TableAffinityStr(Vdbe *v, Table *pTab)
	{
		/* The first time a column affinity string for a particular table
		** is required, it is allocated and populated here. It is then 
		** stored as a member of the Table structure for subsequent use.
		**
		** The column affinity string will eventually be deleted by
		** sqlite3DeleteTable() when the Table structure itself is cleaned up.
		*/
		if( !pTab->zColAff ){
			char *zColAff;
			int i;
			sqlite3 *db = sqlite3VdbeDb(v);

			zColAff = (char *)sqlite3DbMallocRaw(0, pTab->nCol+1);
			if( !zColAff ){
				db->mallocFailed = 1;
				return;
			}

			for(i=0; i<pTab->nCol; i++){
				zColAff[i] = pTab->aCol[i].affinity;
			}
			zColAff[pTab->nCol] = '\0';

			pTab->zColAff = zColAff;
		}

		sqlite3VdbeChangeP4(v, -1, pTab->zColAff, P4_TRANSIENT);
	}


	static int readsTable(Parse *p, int iStartAddr, int iDb, Table *pTab)
	{
		Vdbe *v = sqlite3GetVdbe(p);
		int i;
		int iEnd = sqlite3VdbeCurrentAddr(v);
#ifndef OMIT_VIRTUALTABLE
		VTable *pVTab = IsVirtual(pTab) ? sqlite3GetVTable(p->db, pTab) : 0;
#endif

		for(i=iStartAddr; i<iEnd; i++){
			VdbeOp *pOp = sqlite3VdbeGetOp(v, i);
			assert( pOp!=0 );
			if( pOp->opcode==OP_OpenRead && pOp->p3==iDb ){
				Index *pIndex;
				int tnum = pOp->p2;
				if( tnum==pTab->tnum ){
					return 1;
				}
				for(pIndex=pTab->pIndex; pIndex; pIndex=pIndex->pNext){
					if( tnum==pIndex->tnum ){
						return 1;
					}
				}
			}
#ifndef OMIT_VIRTUALTABLE
			if( pOp->opcode==OP_VOpen && pOp->p4.pVtab==pVTab ){
				assert( pOp->p4.pVtab!=0 );
				assert( pOp->p4type==P4_VTAB );
				return 1;
			}
#endif
		}
		return 0;
	}

#ifndef OMIT_AUTOINCREMENT

	static int autoIncBegin(Parse *pParse, int iDb, Table *pTab)
	{
		int memId = 0;      /* Register holding maximum rowid */
		if( pTab->tabFlags & TF_Autoincrement ){
			Parse *pToplevel = sqlite3ParseToplevel(pParse);
			AutoincInfo *pInfo;

			pInfo = pToplevel->pAinc;
			while( pInfo && pInfo->pTab!=pTab ){ pInfo = pInfo->pNext; }
			if( pInfo==0 ){
				pInfo = sqlite3DbMallocRaw(pParse->db, sizeof(*pInfo));
				if( pInfo==0 ) return 0;
				pInfo->pNext = pToplevel->pAinc;
				pToplevel->pAinc = pInfo;
				pInfo->pTab = pTab;
				pInfo->iDb = iDb;
				pToplevel->nMem++;                  /* Register to hold name of table */
				pInfo->regCtr = ++pToplevel->nMem;  /* Max rowid register */
				pToplevel->nMem++;                  /* Rowid in sqlite_sequence */
			}
			memId = pInfo->regCtr;
		}
		return memId;
	}

	void sqlite3AutoincrementBegin(Parse *pParse)
	{
		AutoincInfo *p;            /* Information about an AUTOINCREMENT */
		sqlite3 *db = pParse->db;  /* The database connection */
		Db *pDb;                   /* Database only autoinc table */
		int memId;                 /* Register holding max rowid */
		int addr;                  /* A VDBE address */
		Vdbe *v = pParse->pVdbe;   /* VDBE under construction */

		/* This routine is never called during trigger-generation.  It is
		** only called from the top-level */
		assert( pParse->pTriggerTab==0 );
		assert( pParse==sqlite3ParseToplevel(pParse) );

		assert( v );   /* We failed long ago if this is not so */
		for(p = pParse->pAinc; p; p = p->pNext){
			pDb = &db->aDb[p->iDb];
			memId = p->regCtr;
			assert( sqlite3SchemaMutexHeld(db, 0, pDb->pSchema) );
			sqlite3OpenTable(pParse, 0, p->iDb, pDb->pSchema->pSeqTab, OP_OpenRead);
			sqlite3VdbeAddOp3(v, OP_Null, 0, memId, memId+1);
			addr = sqlite3VdbeCurrentAddr(v);
			sqlite3VdbeAddOp4(v, OP_String8, 0, memId-1, 0, p->pTab->zName, 0);
			sqlite3VdbeAddOp2(v, OP_Rewind, 0, addr+9);
			sqlite3VdbeAddOp3(v, OP_Column, 0, 0, memId);
			sqlite3VdbeAddOp3(v, OP_Ne, memId-1, addr+7, memId);
			sqlite3VdbeChangeP5(v, SQLITE_JUMPIFNULL);
			sqlite3VdbeAddOp2(v, OP_Rowid, 0, memId+1);
			sqlite3VdbeAddOp3(v, OP_Column, 0, 1, memId);
			sqlite3VdbeAddOp2(v, OP_Goto, 0, addr+9);
			sqlite3VdbeAddOp2(v, OP_Next, 0, addr+2);
			sqlite3VdbeAddOp2(v, OP_Integer, 0, memId);
			sqlite3VdbeAddOp0(v, OP_Close);
		}
	}

	static void autoIncStep(Parse *pParse, int memId, int regRowid)
	{
		if( memId>0 ){
			sqlite3VdbeAddOp2(pParse->pVdbe, OP_MemMax, memId, regRowid);
		}
	}

	void sqlite3AutoincrementEnd(Parse *pParse)
	{
		AutoincInfo *p;
		Vdbe *v = pParse->pVdbe;
		sqlite3 *db = pParse->db;

		assert( v );
		for(p = pParse->pAinc; p; p = p->pNext){
			Db *pDb = &db->aDb[p->iDb];
			int j1, j2, j3, j4, j5;
			int iRec;
			int memId = p->regCtr;

			iRec = sqlite3GetTempReg(pParse);
			assert( sqlite3SchemaMutexHeld(db, 0, pDb->pSchema) );
			sqlite3OpenTable(pParse, 0, p->iDb, pDb->pSchema->pSeqTab, OP_OpenWrite);
			j1 = sqlite3VdbeAddOp1(v, OP_NotNull, memId+1);
			j2 = sqlite3VdbeAddOp0(v, OP_Rewind);
			j3 = sqlite3VdbeAddOp3(v, OP_Column, 0, 0, iRec);
			j4 = sqlite3VdbeAddOp3(v, OP_Eq, memId-1, 0, iRec);
			sqlite3VdbeAddOp2(v, OP_Next, 0, j3);
			sqlite3VdbeJumpHere(v, j2);
			sqlite3VdbeAddOp2(v, OP_NewRowid, 0, memId+1);
			j5 = sqlite3VdbeAddOp0(v, OP_Goto);
			sqlite3VdbeJumpHere(v, j4);
			sqlite3VdbeAddOp2(v, OP_Rowid, 0, memId+1);
			sqlite3VdbeJumpHere(v, j1);
			sqlite3VdbeJumpHere(v, j5);
			sqlite3VdbeAddOp3(v, OP_MakeRecord, memId-1, 2, iRec);
			sqlite3VdbeAddOp3(v, OP_Insert, 0, iRec, memId+1);
			sqlite3VdbeChangeP5(v, OPFLAG_APPEND);
			sqlite3VdbeAddOp0(v, OP_Close);
			sqlite3ReleaseTempReg(pParse, iRec);
		}
	}
#else
#define AutoIncBegin(A,B,C) (0)
#define AutoIncStep(A,B,C)
#endif

	int sqlite3CodeCoroutine(Parse *pParse, Select *pSelect, SelectDest *pDest){
		int regYield;       /* Register holding co-routine entry-point */
		int regEof;         /* Register holding co-routine completion flag */
		int addrTop;        /* Top of the co-routine */
		int j1;             /* Jump instruction */
		int rc;             /* Result code */
		Vdbe *v;            /* VDBE under construction */

		regYield = ++pParse->nMem;
		regEof = ++pParse->nMem;
		v = sqlite3GetVdbe(pParse);
		addrTop = sqlite3VdbeCurrentAddr(v);
		sqlite3VdbeAddOp2(v, OP_Integer, addrTop+2, regYield); /* X <- A */
		VdbeComment((v, "Co-routine entry point"));
		sqlite3VdbeAddOp2(v, OP_Integer, 0, regEof);           /* EOF <- 0 */
		VdbeComment((v, "Co-routine completion flag"));
		sqlite3SelectDestInit(pDest, SRT_Coroutine, regYield);
		j1 = sqlite3VdbeAddOp2(v, OP_Goto, 0, 0);
		rc = sqlite3Select(pParse, pSelect, pDest);
		assert( pParse->nErr==0 || rc );
		if( pParse->db->mallocFailed && rc==SQLITE_OK ) rc = SQLITE_NOMEM;
		if( rc ) return rc;
		sqlite3VdbeAddOp2(v, OP_Integer, 1, regEof);            /* EOF <- 1 */
		sqlite3VdbeAddOp1(v, OP_Yield, regYield);   /* yield X */
		sqlite3VdbeAddOp2(v, OP_Halt, SQLITE_INTERNAL, OE_Abort);
		VdbeComment((v, "End of coroutine"));
		sqlite3VdbeJumpHere(v, j1);                             /* label B: */
		return rc;
	}



	// Forward declaration
	static int XferOptimization(Parse *parse, Table *dest, Select *select, int onError, int dbDestId);

	void sqlite3Insert(Parse *pParse, SrcList *pTabList, ExprList *pList, Select *pSelect, IdList *pColumn, int onError)
	{
		sqlite3 *db;          /* The main database structure */
		Table *pTab;          /* The table to insert into.  aka TABLE */
		char *zTab;           /* Name of the table into which we are inserting */
		const char *zDb;      /* Name of the database holding this table */
		int i, j, idx;        /* Loop counters */
		Vdbe *v;              /* Generate code into this virtual machine */
		Index *pIdx;          /* For looping over indices of the table */
		int nColumn;          /* Number of columns in the data */
		int nHidden = 0;      /* Number of hidden columns if TABLE is virtual */
		int baseCur = 0;      /* VDBE Cursor number for pTab */
		int keyColumn = -1;   /* Column that is the INTEGER PRIMARY KEY */
		int endOfLoop;        /* Label for the end of the insertion loop */
		int useTempTable = 0; /* Store SELECT results in intermediate table */
		int srcTab = 0;       /* Data comes from this temporary cursor if >=0 */
		int addrInsTop = 0;   /* Jump to label "D" */
		int addrCont = 0;     /* Top of insert loop. Label "C" in templates 3 and 4 */
		int addrSelect = 0;   /* Address of coroutine that implements the SELECT */
		SelectDest dest;      /* Destination for SELECT on rhs of INSERT */
		int iDb;              /* Index of database holding TABLE */
		Db *pDb;              /* The database containing table being inserted into */
		int appendFlag = 0;   /* True if the insert is likely to be an append */

		/* Register allocations */
		int regFromSelect = 0;/* Base register for data coming from SELECT */
		int regAutoinc = 0;   /* Register holding the AUTOINCREMENT counter */
		int regRowCount = 0;  /* Memory cell used for the row counter */
		int regIns;           /* Block of regs holding rowid+data being inserted */
		int regRowid;         /* registers holding insert rowid */
		int regData;          /* register holding first column to insert */
		int regEof = 0;       /* Register recording end of SELECT data */
		int *aRegIdx = 0;     /* One register allocated to each index */

#ifndef OMIT_TRIGGER
		int isView;                 /* True if attempting to insert into a view */
		Trigger *pTrigger;          /* List of triggers on pTab, if required */
		int tmask;                  /* Mask of trigger times */
#endif

		db = pParse->db;
		memset(&dest, 0, sizeof(dest));
		if( pParse->nErr || db->mallocFailed ){
			goto insert_cleanup;
		}

		/* Locate the table into which we will be inserting new information.
		*/
		assert( pTabList->nSrc==1 );
		zTab = pTabList->a[0].zName;
		if( NEVER(zTab==0) ) goto insert_cleanup;
		pTab = sqlite3SrcListLookup(pParse, pTabList);
		if( pTab==0 ){
			goto insert_cleanup;
		}
		iDb = sqlite3SchemaToIndex(db, pTab->pSchema);
		assert( iDb<db->nDb );
		pDb = &db->aDb[iDb];
		zDb = pDb->zName;
		if( sqlite3AuthCheck(pParse, SQLITE_INSERT, pTab->zName, 0, zDb) ){
			goto insert_cleanup;
		}

		/* Figure out if we have any triggers and if the table being
		** inserted into is a view
		*/
#ifndef OMIT_TRIGGER
		pTrigger = sqlite3TriggersExist(pParse, pTab, TK_INSERT, 0, &tmask);
		isView = pTab->pSelect!=0;
#else
# define pTrigger 0
# define tmask 0
# define isView 0
#endif
#ifdef OMIT_VIEW
# undef isView
# define isView 0
#endif
		assert( (pTrigger && tmask) || (pTrigger==0 && tmask==0) );

		/* If pTab is really a view, make sure it has been initialized.
		** ViewGetColumnNames() is a no-op if pTab is not a view (or virtual 
		** module table).
		*/
		if( sqlite3ViewGetColumnNames(pParse, pTab) ){
			goto insert_cleanup;
		}

		/* Ensure that:
		*  (a) the table is not read-only, 
		*  (b) that if it is a view then ON INSERT triggers exist
		*/
		if( sqlite3IsReadOnly(pParse, pTab, tmask) ){
			goto insert_cleanup;
		}

		/* Allocate a VDBE
		*/
		v = sqlite3GetVdbe(pParse);
		if( v==0 ) goto insert_cleanup;
		if( pParse->nested==0 ) sqlite3VdbeCountChanges(v);
		sqlite3BeginWriteOperation(pParse, pSelect || pTrigger, iDb);

#ifndef OMIT_XFER_OPT
		/* If the statement is of the form
		**
		**       INSERT INTO <table1> SELECT * FROM <table2>;
		**
		** Then special optimizations can be applied that make the transfer
		** very fast and which reduce fragmentation of indices.
		**
		** This is the 2nd template.
		*/
		if( pColumn==0 && xferOptimization(pParse, pTab, pSelect, onError, iDb) ){
			assert( !pTrigger );
			assert( pList==0 );
			goto insert_end;
		}
#endif

		/* If this is an AUTOINCREMENT table, look up the sequence number in the
		** sqlite_sequence table and store it in memory cell regAutoinc.
		*/
		regAutoinc = autoIncBegin(pParse, iDb, pTab);

		/* Figure out how many columns of data are supplied.  If the data
		** is coming from a SELECT statement, then generate a co-routine that
		** produces a single row of the SELECT on each invocation.  The
		** co-routine is the common header to the 3rd and 4th templates.
		*/
		if( pSelect ){
			/* Data is coming from a SELECT.  Generate a co-routine to run that
			** SELECT. */
			int rc = sqlite3CodeCoroutine(pParse, pSelect, &dest);
			if( rc ) goto insert_cleanup;

			regEof = dest.iSDParm + 1;
			regFromSelect = dest.iSdst;
			assert( pSelect->pEList );
			nColumn = pSelect->pEList->nExpr;
			assert( dest.nSdst==nColumn );

			/* Set useTempTable to TRUE if the result of the SELECT statement
			** should be written into a temporary table (template 4).  Set to
			** FALSE if each* row of the SELECT can be written directly into
			** the destination table (template 3).
			**
			** A temp table must be used if the table being updated is also one
			** of the tables being read by the SELECT statement.  Also use a 
			** temp table in the case of row triggers.
			*/
			if( pTrigger || readsTable(pParse, addrSelect, iDb, pTab) ){
				useTempTable = 1;
			}

			if( useTempTable ){
				/* Invoke the coroutine to extract information from the SELECT
				** and add it to a transient table srcTab.  The code generated
				** here is from the 4th template:
				**
				**      B: open temp table
				**      L: yield X
				**         if EOF goto M
				**         insert row from R..R+n into temp table
				**         goto L
				**      M: ...
				*/
				int regRec;          /* Register to hold packed record */
				int regTempRowid;    /* Register to hold temp table ROWID */
				int addrTop;         /* Label "L" */
				int addrIf;          /* Address of jump to M */

				srcTab = pParse->nTab++;
				regRec = sqlite3GetTempReg(pParse);
				regTempRowid = sqlite3GetTempReg(pParse);
				sqlite3VdbeAddOp2(v, OP_OpenEphemeral, srcTab, nColumn);
				addrTop = sqlite3VdbeAddOp1(v, OP_Yield, dest.iSDParm);
				addrIf = sqlite3VdbeAddOp1(v, OP_If, regEof);
				sqlite3VdbeAddOp3(v, OP_MakeRecord, regFromSelect, nColumn, regRec);
				sqlite3VdbeAddOp2(v, OP_NewRowid, srcTab, regTempRowid);
				sqlite3VdbeAddOp3(v, OP_Insert, srcTab, regRec, regTempRowid);
				sqlite3VdbeAddOp2(v, OP_Goto, 0, addrTop);
				sqlite3VdbeJumpHere(v, addrIf);
				sqlite3ReleaseTempReg(pParse, regRec);
				sqlite3ReleaseTempReg(pParse, regTempRowid);
			}
		}else{
			/* This is the case if the data for the INSERT is coming from a VALUES
			** clause
			*/
			NameContext sNC;
			memset(&sNC, 0, sizeof(sNC));
			sNC.pParse = pParse;
			srcTab = -1;
			assert( useTempTable==0 );
			nColumn = pList ? pList->nExpr : 0;
			for(i=0; i<nColumn; i++){
				if( sqlite3ResolveExprNames(&sNC, pList->a[i].pExpr) ){
					goto insert_cleanup;
				}
			}
		}

		/* Make sure the number of columns in the source data matches the number
		** of columns to be inserted into the table.
		*/
		if( IsVirtual(pTab) ){
			for(i=0; i<pTab->nCol; i++){
				nHidden += (IsHiddenColumn(&pTab->aCol[i]) ? 1 : 0);
			}
		}
		if( pColumn==0 && nColumn && nColumn!=(pTab->nCol-nHidden) ){
			sqlite3ErrorMsg(pParse, 
				"table %S has %d columns but %d values were supplied",
				pTabList, 0, pTab->nCol-nHidden, nColumn);
			goto insert_cleanup;
		}
		if( pColumn!=0 && nColumn!=pColumn->nId ){
			sqlite3ErrorMsg(pParse, "%d values for %d columns", nColumn, pColumn->nId);
			goto insert_cleanup;
		}

		/* If the INSERT statement included an IDLIST term, then make sure
		** all elements of the IDLIST really are columns of the table and 
		** remember the column indices.
		**
		** If the table has an INTEGER PRIMARY KEY column and that column
		** is named in the IDLIST, then record in the keyColumn variable
		** the index into IDLIST of the primary key column.  keyColumn is
		** the index of the primary key as it appears in IDLIST, not as
		** is appears in the original table.  (The index of the primary
		** key in the original table is pTab->iPKey.)
		*/
		if( pColumn ){
			for(i=0; i<pColumn->nId; i++){
				pColumn->a[i].idx = -1;
			}
			for(i=0; i<pColumn->nId; i++){
				for(j=0; j<pTab->nCol; j++){
					if( sqlite3StrICmp(pColumn->a[i].zName, pTab->aCol[j].zName)==0 ){
						pColumn->a[i].idx = j;
						if( j==pTab->iPKey ){
							keyColumn = i;
						}
						break;
					}
				}
				if( j>=pTab->nCol ){
					if( sqlite3IsRowid(pColumn->a[i].zName) ){
						keyColumn = i;
					}else{
						sqlite3ErrorMsg(pParse, "table %S has no column named %s",
							pTabList, 0, pColumn->a[i].zName);
						pParse->checkSchema = 1;
						goto insert_cleanup;
					}
				}
			}
		}

		/* If there is no IDLIST term but the table has an integer primary
		** key, the set the keyColumn variable to the primary key column index
		** in the original table definition.
		*/
		if( pColumn==0 && nColumn>0 ){
			keyColumn = pTab->iPKey;
		}

		/* Initialize the count of rows to be inserted
		*/
		if( db->flags & SQLITE_CountRows ){
			regRowCount = ++pParse->nMem;
			sqlite3VdbeAddOp2(v, OP_Integer, 0, regRowCount);
		}

		/* If this is not a view, open the table and and all indices */
		if( !isView ){
			int nIdx;

			baseCur = pParse->nTab;
			nIdx = sqlite3OpenTableAndIndices(pParse, pTab, baseCur, OP_OpenWrite);
			aRegIdx = sqlite3DbMallocRaw(db, sizeof(int)*(nIdx+1));
			if( aRegIdx==0 ){
				goto insert_cleanup;
			}
			for(i=0; i<nIdx; i++){
				aRegIdx[i] = ++pParse->nMem;
			}
		}

		/* This is the top of the main insertion loop */
		if( useTempTable ){
			/* This block codes the top of loop only.  The complete loop is the
			** following pseudocode (template 4):
			**
			**         rewind temp table
			**      C: loop over rows of intermediate table
			**           transfer values form intermediate table into <table>
			**         end loop
			**      D: ...
			*/
			addrInsTop = sqlite3VdbeAddOp1(v, OP_Rewind, srcTab);
			addrCont = sqlite3VdbeCurrentAddr(v);
		}else if( pSelect ){
			/* This block codes the top of loop only.  The complete loop is the
			** following pseudocode (template 3):
			**
			**      C: yield X
			**         if EOF goto D
			**         insert the select result into <table> from R..R+n
			**         goto C
			**      D: ...
			*/
			addrCont = sqlite3VdbeAddOp1(v, OP_Yield, dest.iSDParm);
			addrInsTop = sqlite3VdbeAddOp1(v, OP_If, regEof);
		}

		/* Allocate registers for holding the rowid of the new row,
		** the content of the new row, and the assemblied row record.
		*/
		regRowid = regIns = pParse->nMem+1;
		pParse->nMem += pTab->nCol + 1;
		if( IsVirtual(pTab) ){
			regRowid++;
			pParse->nMem++;
		}
		regData = regRowid+1;

		/* Run the BEFORE and INSTEAD OF triggers, if there are any
		*/
		endOfLoop = sqlite3VdbeMakeLabel(v);
		if( tmask & TRIGGER_BEFORE ){
			int regCols = sqlite3GetTempRange(pParse, pTab->nCol+1);

			/* build the NEW.* reference row.  Note that if there is an INTEGER
			** PRIMARY KEY into which a NULL is being inserted, that NULL will be
			** translated into a unique ID for the row.  But on a BEFORE trigger,
			** we do not know what the unique ID will be (because the insert has
			** not happened yet) so we substitute a rowid of -1
			*/
			if( keyColumn<0 ){
				sqlite3VdbeAddOp2(v, OP_Integer, -1, regCols);
			}else{
				int j1;
				if( useTempTable ){
					sqlite3VdbeAddOp3(v, OP_Column, srcTab, keyColumn, regCols);
				}else{
					assert( pSelect==0 );  /* Otherwise useTempTable is true */
					sqlite3ExprCode(pParse, pList->a[keyColumn].pExpr, regCols);
				}
				j1 = sqlite3VdbeAddOp1(v, OP_NotNull, regCols);
				sqlite3VdbeAddOp2(v, OP_Integer, -1, regCols);
				sqlite3VdbeJumpHere(v, j1);
				sqlite3VdbeAddOp1(v, OP_MustBeInt, regCols);
			}

			/* Cannot have triggers on a virtual table. If it were possible,
			** this block would have to account for hidden column.
			*/
			assert( !IsVirtual(pTab) );

			/* Create the new column data
			*/
			for(i=0; i<pTab->nCol; i++){
				if( pColumn==0 ){
					j = i;
				}else{
					for(j=0; j<pColumn->nId; j++){
						if( pColumn->a[j].idx==i ) break;
					}
				}
				if( (!useTempTable && !pList) || (pColumn && j>=pColumn->nId) ){
					sqlite3ExprCode(pParse, pTab->aCol[i].pDflt, regCols+i+1);
				}else if( useTempTable ){
					sqlite3VdbeAddOp3(v, OP_Column, srcTab, j, regCols+i+1); 
				}else{
					assert( pSelect==0 ); /* Otherwise useTempTable is true */
					sqlite3ExprCodeAndCache(pParse, pList->a[j].pExpr, regCols+i+1);
				}
			}

			/* If this is an INSERT on a view with an INSTEAD OF INSERT trigger,
			** do not attempt any conversions before assembling the record.
			** If this is a real table, attempt conversions as required by the
			** table column affinities.
			*/
			if( !isView ){
				sqlite3VdbeAddOp2(v, OP_Affinity, regCols+1, pTab->nCol);
				sqlite3TableAffinityStr(v, pTab);
			}

			/* Fire BEFORE or INSTEAD OF triggers */
			sqlite3CodeRowTrigger(pParse, pTrigger, TK_INSERT, 0, TRIGGER_BEFORE, 
				pTab, regCols-pTab->nCol-1, onError, endOfLoop);

			sqlite3ReleaseTempRange(pParse, regCols, pTab->nCol+1);
		}

		/* Push the record number for the new entry onto the stack.  The
		** record number is a randomly generate integer created by NewRowid
		** except when the table has an INTEGER PRIMARY KEY column, in which
		** case the record number is the same as that column. 
		*/
		if( !isView ){
			if( IsVirtual(pTab) ){
				/* The row that the VUpdate opcode will delete: none */
				sqlite3VdbeAddOp2(v, OP_Null, 0, regIns);
			}
			if( keyColumn>=0 ){
				if( useTempTable ){
					sqlite3VdbeAddOp3(v, OP_Column, srcTab, keyColumn, regRowid);
				}else if( pSelect ){
					sqlite3VdbeAddOp2(v, OP_SCopy, regFromSelect+keyColumn, regRowid);
				}else{
					VdbeOp *pOp;
					sqlite3ExprCode(pParse, pList->a[keyColumn].pExpr, regRowid);
					pOp = sqlite3VdbeGetOp(v, -1);
					if( ALWAYS(pOp) && pOp->opcode==OP_Null && !IsVirtual(pTab) ){
						appendFlag = 1;
						pOp->opcode = OP_NewRowid;
						pOp->p1 = baseCur;
						pOp->p2 = regRowid;
						pOp->p3 = regAutoinc;
					}
				}
				/* If the PRIMARY KEY expression is NULL, then use OP_NewRowid
				** to generate a unique primary key value.
				*/
				if( !appendFlag ){
					int j1;
					if( !IsVirtual(pTab) ){
						j1 = sqlite3VdbeAddOp1(v, OP_NotNull, regRowid);
						sqlite3VdbeAddOp3(v, OP_NewRowid, baseCur, regRowid, regAutoinc);
						sqlite3VdbeJumpHere(v, j1);
					}else{
						j1 = sqlite3VdbeCurrentAddr(v);
						sqlite3VdbeAddOp2(v, OP_IsNull, regRowid, j1+2);
					}
					sqlite3VdbeAddOp1(v, OP_MustBeInt, regRowid);
				}
			}else if( IsVirtual(pTab) ){
				sqlite3VdbeAddOp2(v, OP_Null, 0, regRowid);
			}else{
				sqlite3VdbeAddOp3(v, OP_NewRowid, baseCur, regRowid, regAutoinc);
				appendFlag = 1;
			}
			autoIncStep(pParse, regAutoinc, regRowid);

			/* Push onto the stack, data for all columns of the new entry, beginning
			** with the first column.
			*/
			nHidden = 0;
			for(i=0; i<pTab->nCol; i++){
				int iRegStore = regRowid+1+i;
				if( i==pTab->iPKey ){
					/* The value of the INTEGER PRIMARY KEY column is always a NULL.
					** Whenever this column is read, the record number will be substituted
					** in its place.  So will fill this column with a NULL to avoid
					** taking up data space with information that will never be used. */
					sqlite3VdbeAddOp2(v, OP_Null, 0, iRegStore);
					continue;
				}
				if( pColumn==0 ){
					if( IsHiddenColumn(&pTab->aCol[i]) ){
						assert( IsVirtual(pTab) );
						j = -1;
						nHidden++;
					}else{
						j = i - nHidden;
					}
				}else{
					for(j=0; j<pColumn->nId; j++){
						if( pColumn->a[j].idx==i ) break;
					}
				}
				if( j<0 || nColumn==0 || (pColumn && j>=pColumn->nId) ){
					sqlite3ExprCode(pParse, pTab->aCol[i].pDflt, iRegStore);
				}else if( useTempTable ){
					sqlite3VdbeAddOp3(v, OP_Column, srcTab, j, iRegStore); 
				}else if( pSelect ){
					sqlite3VdbeAddOp2(v, OP_SCopy, regFromSelect+j, iRegStore);
				}else{
					sqlite3ExprCode(pParse, pList->a[j].pExpr, iRegStore);
				}
			}

			/* Generate code to check constraints and generate index keys and
			** do the insertion.
			*/
#ifndef OMIT_VIRTUALTABLE
			if( IsVirtual(pTab) ){
				const char *pVTab = (const char *)sqlite3GetVTable(db, pTab);
				sqlite3VtabMakeWritable(pParse, pTab);
				sqlite3VdbeAddOp4(v, OP_VUpdate, 1, pTab->nCol+2, regIns, pVTab, P4_VTAB);
				sqlite3VdbeChangeP5(v, onError==OE_Default ? OE_Abort : onError);
				sqlite3MayAbort(pParse);
			}else
#endif
			{
				int isReplace;    /* Set to true if constraints may cause a replace */
				sqlite3GenerateConstraintChecks(pParse, pTab, baseCur, regIns, aRegIdx,
					keyColumn>=0, 0, onError, endOfLoop, &isReplace
					);
				sqlite3FkCheck(pParse, pTab, 0, regIns);
				sqlite3CompleteInsertion(
					pParse, pTab, baseCur, regIns, aRegIdx, 0, appendFlag, isReplace==0
					);
			}
		}

		/* Update the count of rows that are inserted
		*/
		if( (db->flags & SQLITE_CountRows)!=0 ){
			sqlite3VdbeAddOp2(v, OP_AddImm, regRowCount, 1);
		}

		if( pTrigger ){
			/* Code AFTER triggers */
			sqlite3CodeRowTrigger(pParse, pTrigger, TK_INSERT, 0, TRIGGER_AFTER, 
				pTab, regData-2-pTab->nCol, onError, endOfLoop);
		}

		/* The bottom of the main insertion loop, if the data source
		** is a SELECT statement.
		*/
		sqlite3VdbeResolveLabel(v, endOfLoop);
		if( useTempTable ){
			sqlite3VdbeAddOp2(v, OP_Next, srcTab, addrCont);
			sqlite3VdbeJumpHere(v, addrInsTop);
			sqlite3VdbeAddOp1(v, OP_Close, srcTab);
		}else if( pSelect ){
			sqlite3VdbeAddOp2(v, OP_Goto, 0, addrCont);
			sqlite3VdbeJumpHere(v, addrInsTop);
		}

		if( !IsVirtual(pTab) && !isView ){
			/* Close all tables opened */
			sqlite3VdbeAddOp1(v, OP_Close, baseCur);
			for(idx=1, pIdx=pTab->pIndex; pIdx; pIdx=pIdx->pNext, idx++){
				sqlite3VdbeAddOp1(v, OP_Close, idx+baseCur);
			}
		}

insert_end:
		/* Update the sqlite_sequence table by storing the content of the
		** maximum rowid counter values recorded while inserting into
		** autoincrement tables.
		*/
		if( pParse->nested==0 && pParse->pTriggerTab==0 ){
			sqlite3AutoincrementEnd(pParse);
		}

		/*
		** Return the number of rows inserted. If this routine is 
		** generating code because of a call to sqlite3NestedParse(), do not
		** invoke the callback function.
		*/
		if( (db->flags&SQLITE_CountRows) && !pParse->nested && !pParse->pTriggerTab ){
			sqlite3VdbeAddOp2(v, OP_ResultRow, regRowCount, 1);
			sqlite3VdbeSetNumCols(v, 1);
			sqlite3VdbeSetColName(v, 0, COLNAME_NAME, "rows inserted", SQLITE_STATIC);
		}

insert_cleanup:
		sqlite3SrcListDelete(db, pTabList);
		sqlite3ExprListDelete(db, pList);
		sqlite3SelectDelete(db, pSelect);
		sqlite3IdListDelete(db, pColumn);
		sqlite3DbFree(db, aRegIdx);
	}

	/* Make sure "isView" and other macros defined above are undefined. Otherwise
	** thely may interfere with compilation of other functions in this file
	** (or in another file, if this file becomes part of the amalgamation).  */
#ifdef isView
#undef isView
#endif
#ifdef pTrigger
#undef pTrigger
#endif
#ifdef tmask
#undef tmask
#endif

	void sqlite3GenerateConstraintChecks(Parse *pParse, Table *pTab, int baseCur, int regRowid, int *aRegIdx, int rowidChng, int isUpdate, int overrideError, int ignoreDest, int *pbMayReplace)
	{
		int i;              /* loop counter */
		Vdbe *v;            /* VDBE under constrution */
		int nCol;           /* Number of columns */
		int onError;        /* Conflict resolution strategy */
		int j1;             /* Addresss of jump instruction */
		int j2 = 0, j3;     /* Addresses of jump instructions */
		int regData;        /* Register containing first data column */
		int iCur;           /* Table cursor number */
		Index *pIdx;         /* Pointer to one of the indices */
		sqlite3 *db;         /* Database connection */
		int seenReplace = 0; /* True if REPLACE is used to resolve INT PK conflict */
		int regOldRowid = (rowidChng && isUpdate) ? rowidChng : regRowid;

		db = pParse->db;
		v = sqlite3GetVdbe(pParse);
		assert( v!=0 );
		assert( pTab->pSelect==0 );  /* This table is not a VIEW */
		nCol = pTab->nCol;
		regData = regRowid + 1;

		/* Test all NOT NULL constraints.
		*/
		for(i=0; i<nCol; i++){
			if( i==pTab->iPKey ){
				continue;
			}
			onError = pTab->aCol[i].notNull;
			if( onError==OE_None ) continue;
			if( overrideError!=OE_Default ){
				onError = overrideError;
			}else if( onError==OE_Default ){
				onError = OE_Abort;
			}
			if( onError==OE_Replace && pTab->aCol[i].pDflt==0 ){
				onError = OE_Abort;
			}
			assert( onError==OE_Rollback || onError==OE_Abort || onError==OE_Fail
				|| onError==OE_Ignore || onError==OE_Replace );
			switch( onError ){
			case OE_Abort:
				sqlite3MayAbort(pParse);
			case OE_Rollback:
			case OE_Fail: {
				char *zMsg;
				sqlite3VdbeAddOp3(v, OP_HaltIfNull,
					SQLITE_CONSTRAINT_NOTNULL, onError, regData+i);
				zMsg = sqlite3MPrintf(db, "%s.%s may not be NULL",
					pTab->zName, pTab->aCol[i].zName);
				sqlite3VdbeChangeP4(v, -1, zMsg, P4_DYNAMIC);
				break;
						  }
			case OE_Ignore: {
				sqlite3VdbeAddOp2(v, OP_IsNull, regData+i, ignoreDest);
				break;
							}
			default: {
				assert( onError==OE_Replace );
				j1 = sqlite3VdbeAddOp1(v, OP_NotNull, regData+i);
				sqlite3ExprCode(pParse, pTab->aCol[i].pDflt, regData+i);
				sqlite3VdbeJumpHere(v, j1);
				break;
					 }
			}
		}

		// Test all CHECK constraints
#ifndef OMIT_CHECK
		if( pTab->pCheck && (db->flags & SQLITE_IgnoreChecks)==0 ){
			ExprList *pCheck = pTab->pCheck;
			pParse->ckBase = regData;
			onError = overrideError!=OE_Default ? overrideError : OE_Abort;
			for(i=0; i<pCheck->nExpr; i++){
				int allOk = sqlite3VdbeMakeLabel(v);
				sqlite3ExprIfTrue(pParse, pCheck->a[i].pExpr, allOk, SQLITE_JUMPIFNULL);
				if( onError==OE_Ignore ){
					sqlite3VdbeAddOp2(v, OP_Goto, 0, ignoreDest);
				}else{
					char *zConsName = pCheck->a[i].zName;
					if( onError==OE_Replace ) onError = OE_Abort; /* IMP: R-15569-63625 */
					if( zConsName ){
						zConsName = sqlite3MPrintf(db, "constraint %s failed", zConsName);
					}else{
						zConsName = 0;
					}
					sqlite3HaltConstraint(pParse, SQLITE_CONSTRAINT_CHECK,
						onError, zConsName, P4_DYNAMIC);
				}
				sqlite3VdbeResolveLabel(v, allOk);
			}
		}
#endif

		/* If we have an INTEGER PRIMARY KEY, make sure the primary key
		** of the new record does not previously exist.  Except, if this
		** is an UPDATE and the primary key is not changing, that is OK.
		*/
		if( rowidChng ){
			onError = pTab->keyConf;
			if( overrideError!=OE_Default ){
				onError = overrideError;
			}else if( onError==OE_Default ){
				onError = OE_Abort;
			}

			if( isUpdate ){
				j2 = sqlite3VdbeAddOp3(v, OP_Eq, regRowid, 0, rowidChng);
			}
			j3 = sqlite3VdbeAddOp3(v, OP_NotExists, baseCur, 0, regRowid);
			switch( onError ){
			default: {
				onError = OE_Abort;
				/* Fall thru into the next case */
					 }
			case OE_Rollback:
			case OE_Abort:
			case OE_Fail: {
				sqlite3HaltConstraint(pParse, SQLITE_CONSTRAINT_PRIMARYKEY,
					onError, "PRIMARY KEY must be unique", P4_STATIC);
				break;
						  }
			case OE_Replace: {
				/* If there are DELETE triggers on this table and the
				** recursive-triggers flag is set, call GenerateRowDelete() to
				** remove the conflicting row from the table. This will fire
				** the triggers and remove both the table and index b-tree entries.
				**
				** Otherwise, if there are no triggers or the recursive-triggers
				** flag is not set, but the table has one or more indexes, call 
				** GenerateRowIndexDelete(). This removes the index b-tree entries 
				** only. The table b-tree entry will be replaced by the new entry 
				** when it is inserted.  
				**
				** If either GenerateRowDelete() or GenerateRowIndexDelete() is called,
				** also invoke MultiWrite() to indicate that this VDBE may require
				** statement rollback (if the statement is aborted after the delete
				** takes place). Earlier versions called sqlite3MultiWrite() regardless,
				** but being more selective here allows statements like:
				**
				**   REPLACE INTO t(rowid) VALUES($newrowid)
				**
				** to run without a statement journal if there are no indexes on the
				** table.
				*/
				Trigger *pTrigger = 0;
				if( db->flags&SQLITE_RecTriggers ){
					pTrigger = sqlite3TriggersExist(pParse, pTab, TK_DELETE, 0, 0);
				}
				if( pTrigger || sqlite3FkRequired(pParse, pTab, 0, 0) ){
					sqlite3MultiWrite(pParse);
					sqlite3GenerateRowDelete(
						pParse, pTab, baseCur, regRowid, 0, pTrigger, OE_Replace
						);
				}else if( pTab->pIndex ){
					sqlite3MultiWrite(pParse);
					sqlite3GenerateRowIndexDelete(pParse, pTab, baseCur, 0);
				}
				seenReplace = 1;
				break;
							 }
			case OE_Ignore: {
				assert( seenReplace==0 );
				sqlite3VdbeAddOp2(v, OP_Goto, 0, ignoreDest);
				break;
							}
			}
			sqlite3VdbeJumpHere(v, j3);
			if( isUpdate ){
				sqlite3VdbeJumpHere(v, j2);
			}
		}

		/* Test all UNIQUE constraints by creating entries for each UNIQUE
		** index and making sure that duplicate entries do not already exist.
		** Add the new records to the indices as we go.
		*/
		for(iCur=0, pIdx=pTab->pIndex; pIdx; pIdx=pIdx->pNext, iCur++){
			int regIdx;
			int regR;

			if( aRegIdx[iCur]==0 ) continue;  /* Skip unused indices */

			/* Create a key for accessing the index entry */
			regIdx = sqlite3GetTempRange(pParse, pIdx->nColumn+1);
			for(i=0; i<pIdx->nColumn; i++){
				int idx = pIdx->aiColumn[i];
				if( idx==pTab->iPKey ){
					sqlite3VdbeAddOp2(v, OP_SCopy, regRowid, regIdx+i);
				}else{
					sqlite3VdbeAddOp2(v, OP_SCopy, regData+idx, regIdx+i);
				}
			}
			sqlite3VdbeAddOp2(v, OP_SCopy, regRowid, regIdx+i);
			sqlite3VdbeAddOp3(v, OP_MakeRecord, regIdx, pIdx->nColumn+1, aRegIdx[iCur]);
			sqlite3VdbeChangeP4(v, -1, sqlite3IndexAffinityStr(v, pIdx), P4_TRANSIENT);
			sqlite3ExprCacheAffinityChange(pParse, regIdx, pIdx->nColumn+1);

			/* Find out what action to take in case there is an indexing conflict */
			onError = pIdx->onError;
			if( onError==OE_None ){ 
				sqlite3ReleaseTempRange(pParse, regIdx, pIdx->nColumn+1);
				continue;  /* pIdx is not a UNIQUE index */
			}
			if( overrideError!=OE_Default ){
				onError = overrideError;
			}else if( onError==OE_Default ){
				onError = OE_Abort;
			}
			if( seenReplace ){
				if( onError==OE_Ignore ) onError = OE_Replace;
				else if( onError==OE_Fail ) onError = OE_Abort;
			}

			/* Check to see if the new index entry will be unique */
			regR = sqlite3GetTempReg(pParse);
			sqlite3VdbeAddOp2(v, OP_SCopy, regOldRowid, regR);
			j3 = sqlite3VdbeAddOp4(v, OP_IsUnique, baseCur+iCur+1, 0,
				regR, SQLITE_INT_TO_PTR(regIdx),
				P4_INT32);
			sqlite3ReleaseTempRange(pParse, regIdx, pIdx->nColumn+1);

			/* Generate code that executes if the new index entry is not unique */
			assert( onError==OE_Rollback || onError==OE_Abort || onError==OE_Fail
				|| onError==OE_Ignore || onError==OE_Replace );
			switch( onError ){
			case OE_Rollback:
			case OE_Abort:
			case OE_Fail: {
				int j;
				StrAccum errMsg;
				const char *zSep;
				char *zErr;

				sqlite3StrAccumInit(&errMsg, 0, 0, 200);
				errMsg.db = db;
				zSep = pIdx->nColumn>1 ? "columns " : "column ";
				for(j=0; j<pIdx->nColumn; j++){
					char *zCol = pTab->aCol[pIdx->aiColumn[j]].zName;
					sqlite3StrAccumAppend(&errMsg, zSep, -1);
					zSep = ", ";
					sqlite3StrAccumAppend(&errMsg, zCol, -1);
				}
				sqlite3StrAccumAppend(&errMsg,
					pIdx->nColumn>1 ? " are not unique" : " is not unique", -1);
				zErr = sqlite3StrAccumFinish(&errMsg);
				sqlite3HaltConstraint(pParse, SQLITE_CONSTRAINT_UNIQUE,
					onError, zErr, 0);
				sqlite3DbFree(errMsg.db, zErr);
				break;
						  }
			case OE_Ignore: {
				assert( seenReplace==0 );
				sqlite3VdbeAddOp2(v, OP_Goto, 0, ignoreDest);
				break;
							}
			default: {
				Trigger *pTrigger = 0;
				assert( onError==OE_Replace );
				sqlite3MultiWrite(pParse);
				if( db->flags&SQLITE_RecTriggers ){
					pTrigger = sqlite3TriggersExist(pParse, pTab, TK_DELETE, 0, 0);
				}
				sqlite3GenerateRowDelete(
					pParse, pTab, baseCur, regR, 0, pTrigger, OE_Replace
					);
				seenReplace = 1;
				break;
					 }
			}
			sqlite3VdbeJumpHere(v, j3);
			sqlite3ReleaseTempReg(pParse, regR);
		}

		if( pbMayReplace ){
			*pbMayReplace = seenReplace;
		}
	}

	void sqlite3CompleteInsertion(Parse *pParse, Table *pTab, int baseCur, int regRowid, int *aRegIdx, int isUpdate, int appendBias, int useSeekResult)
	{
		int i;
		Vdbe *v;
		int nIdx;
		Index *pIdx;
		u8 pik_flags;
		int regData;
		int regRec;

		v = sqlite3GetVdbe(pParse);
		assert( v!=0 );
		assert( pTab->pSelect==0 );  /* This table is not a VIEW */
		for(nIdx=0, pIdx=pTab->pIndex; pIdx; pIdx=pIdx->pNext, nIdx++){}
		for(i=nIdx-1; i>=0; i--){
			if( aRegIdx[i]==0 ) continue;
			sqlite3VdbeAddOp2(v, OP_IdxInsert, baseCur+i+1, aRegIdx[i]);
			if( useSeekResult ){
				sqlite3VdbeChangeP5(v, OPFLAG_USESEEKRESULT);
			}
		}
		regData = regRowid + 1;
		regRec = sqlite3GetTempReg(pParse);
		sqlite3VdbeAddOp3(v, OP_MakeRecord, regData, pTab->nCol, regRec);
		sqlite3TableAffinityStr(v, pTab);
		sqlite3ExprCacheAffinityChange(pParse, regData, pTab->nCol);
		if( pParse->nested ){
			pik_flags = 0;
		}else{
			pik_flags = OPFLAG_NCHANGE;
			pik_flags |= (isUpdate?OPFLAG_ISUPDATE:OPFLAG_LASTROWID);
		}
		if( appendBias ){
			pik_flags |= OPFLAG_APPEND;
		}
		if( useSeekResult ){
			pik_flags |= OPFLAG_USESEEKRESULT;
		}
		sqlite3VdbeAddOp3(v, OP_Insert, baseCur, regRec, regRowid);
		if( !pParse->nested ){
			sqlite3VdbeChangeP4(v, -1, pTab->zName, P4_TRANSIENT);
		}
		sqlite3VdbeChangeP5(v, pik_flags);
	}

	int sqlite3OpenTableAndIndices(
		Parse *pParse,   /* Parsing context */
		Table *pTab,     /* Table to be opened */
		int baseCur,     /* Cursor number assigned to the table */
		int op           /* OP_OpenRead or OP_OpenWrite */
		){
			int i;
			int iDb;
			Index *pIdx;
			Vdbe *v;

			if( IsVirtual(pTab) ) return 0;
			iDb = sqlite3SchemaToIndex(pParse->db, pTab->pSchema);
			v = sqlite3GetVdbe(pParse);
			assert( v!=0 );
			sqlite3OpenTable(pParse, baseCur, iDb, pTab, op);
			for(i=1, pIdx=pTab->pIndex; pIdx; pIdx=pIdx->pNext, i++){
				KeyInfo *pKey = sqlite3IndexKeyinfo(pParse, pIdx);
				assert( pIdx->pSchema==pTab->pSchema );
				sqlite3VdbeAddOp4(v, op, i+baseCur, pIdx->tnum, iDb,
					(char*)pKey, P4_KEYINFO_HANDOFF);
				VdbeComment((v, "%s", pIdx->zName));
			}
			if( pParse->nTab<baseCur+i ){
				pParse->nTab = baseCur+i;
			}
			return i-1;
	}


#ifdef TEST
	int sqlite3_xferopt_count;
#endif


#ifndef OMIT_XFER_OPT
	static int xferCompatibleCollation(const char *z1, const char *z2){
		if( z1==0 ){
			return z2==0;
		}
		if( z2==0 ){
			return 0;
		}
		return sqlite3StrICmp(z1, z2)==0;
	}

	static int XferCompatibleIndex(Index *pDest, Index *pSrc){
		int i;
		assert( pDest && pSrc );
		assert( pDest->pTable!=pSrc->pTable );
		if( pDest->nColumn!=pSrc->nColumn ){
			return 0;   /* Different number of columns */
		}
		if( pDest->onError!=pSrc->onError ){
			return 0;   /* Different conflict resolution strategies */
		}
		for(i=0; i<pSrc->nColumn; i++){
			if( pSrc->aiColumn[i]!=pDest->aiColumn[i] ){
				return 0;   /* Different columns indexed */
			}
			if( pSrc->aSortOrder[i]!=pDest->aSortOrder[i] ){
				return 0;   /* Different sort orders */
			}
			if( !xferCompatibleCollation(pSrc->azColl[i],pDest->azColl[i]) ){
				return 0;   /* Different collating sequences */
			}
		}

		/* If no test above fails then the indices must be compatible */
		return 1;
	}

	static int XferOptimization(Parse *parse, Table *dest, Select *select, int onError, int dbDestId)
	{
		ExprList *pEList;                /* The result set of the SELECT */
		Table *pSrc;                     /* The table in the FROM clause of SELECT */
		Index *pSrcIdx, *pDestIdx;       /* Source and destination indices */
		struct SrcList_item *pItem;      /* An element of pSelect->pSrc */
		int i;                           /* Loop counter */
		int iDbSrc;                      /* The database of pSrc */
		int iSrc, iDest;                 /* Cursors from source and destination */
		int addr1, addr2;                /* Loop addresses */
		int emptyDestTest;               /* Address of test for empty pDest */
		int emptySrcTest;                /* Address of test for empty pSrc */
		Vdbe *v;                         /* The VDBE we are building */
		KeyInfo *pKey;                   /* Key information for an index */
		int regAutoinc;                  /* Memory register used by AUTOINC */
		int destHasUniqueIdx = 0;        /* True if pDest has a UNIQUE index */
		int regData, regRowid;           /* Registers holding data and rowid */

		if( pSelect==0 ){
			return 0;   /* Must be of the form  INSERT INTO ... SELECT ... */
		}
		if( sqlite3TriggerList(pParse, pDest) ){
			return 0;   /* tab1 must not have triggers */
		}
#ifndef OMIT_VIRTUALTABLE
		if( pDest->tabFlags & TF_Virtual ){
			return 0;   /* tab1 must not be a virtual table */
		}
#endif
		if( onError==OE_Default ){
			if( pDest->iPKey>=0 ) onError = pDest->keyConf;
			if( onError==OE_Default ) onError = OE_Abort;
		}
		assert(pSelect->pSrc);   /* allocated even if there is no FROM clause */
		if( pSelect->pSrc->nSrc!=1 ){
			return 0;   /* FROM clause must have exactly one term */
		}
		if( pSelect->pSrc->a[0].pSelect ){
			return 0;   /* FROM clause cannot contain a subquery */
		}
		if( pSelect->pWhere ){
			return 0;   /* SELECT may not have a WHERE clause */
		}
		if( pSelect->pOrderBy ){
			return 0;   /* SELECT may not have an ORDER BY clause */
		}
		/* Do not need to test for a HAVING clause.  If HAVING is present but
		** there is no ORDER BY, we will get an error. */
		if( pSelect->pGroupBy ){
			return 0;   /* SELECT may not have a GROUP BY clause */
		}
		if( pSelect->pLimit ){
			return 0;   /* SELECT may not have a LIMIT clause */
		}
		assert( pSelect->pOffset==0 );  /* Must be so if pLimit==0 */
		if( pSelect->pPrior ){
			return 0;   /* SELECT may not be a compound query */
		}
		if( pSelect->selFlags & SF_Distinct ){
			return 0;   /* SELECT may not be DISTINCT */
		}
		pEList = pSelect->pEList;
		assert( pEList!=0 );
		if( pEList->nExpr!=1 ){
			return 0;   /* The result set must have exactly one column */
		}
		assert( pEList->a[0].pExpr );
		if( pEList->a[0].pExpr->op!=TK_ALL ){
			return 0;   /* The result set must be the special operator "*" */
		}

		/* At this point we have established that the statement is of the
		** correct syntactic form to participate in this optimization.  Now
		** we have to check the semantics.
		*/
		pItem = pSelect->pSrc->a;
		pSrc = sqlite3LocateTableItem(pParse, 0, pItem);
		if( pSrc==0 ){
			return 0;   /* FROM clause does not contain a real table */
		}
		if( pSrc==pDest ){
			return 0;   /* tab1 and tab2 may not be the same table */
		}
#ifndef OMIT_VIRTUALTABLE
		if( pSrc->tabFlags & TF_Virtual ){
			return 0;   /* tab2 must not be a virtual table */
		}
#endif
		if( pSrc->pSelect ){
			return 0;   /* tab2 may not be a view */
		}
		if( pDest->nCol!=pSrc->nCol ){
			return 0;   /* Number of columns must be the same in tab1 and tab2 */
		}
		if( pDest->iPKey!=pSrc->iPKey ){
			return 0;   /* Both tables must have the same INTEGER PRIMARY KEY */
		}
		for(i=0; i<pDest->nCol; i++){
			if( pDest->aCol[i].affinity!=pSrc->aCol[i].affinity ){
				return 0;    /* Affinity must be the same on all columns */
			}
			if( !xferCompatibleCollation(pDest->aCol[i].zColl, pSrc->aCol[i].zColl) ){
				return 0;    /* Collating sequence must be the same on all columns */
			}
			if( pDest->aCol[i].notNull && !pSrc->aCol[i].notNull ){
				return 0;    /* tab2 must be NOT NULL if tab1 is */
			}
		}
		for(pDestIdx=pDest->pIndex; pDestIdx; pDestIdx=pDestIdx->pNext){
			if( pDestIdx->onError!=OE_None ){
				destHasUniqueIdx = 1;
			}
			for(pSrcIdx=pSrc->pIndex; pSrcIdx; pSrcIdx=pSrcIdx->pNext){
				if( xferCompatibleIndex(pDestIdx, pSrcIdx) ) break;
			}
			if( pSrcIdx==0 ){
				return 0;    /* pDestIdx has no corresponding index in pSrc */
			}
		}
#ifndef OMIT_CHECK
		if( pDest->pCheck && sqlite3ExprListCompare(pSrc->pCheck, pDest->pCheck) ){
			return 0;   /* Tables have different CHECK constraints.  Ticket #2252 */
		}
#endif
#ifndef OMIT_FOREIGN_KEY
		/* Disallow the transfer optimization if the destination table constains
		** any foreign key constraints.  This is more restrictive than necessary.
		** But the main beneficiary of the transfer optimization is the VACUUM 
		** command, and the VACUUM command disables foreign key constraints.  So
		** the extra complication to make this rule less restrictive is probably
		** not worth the effort.  Ticket [6284df89debdfa61db8073e062908af0c9b6118e]
		*/
		if( (pParse->db->flags & SQLITE_ForeignKeys)!=0 && pDest->pFKey!=0 ){
			return 0;
		}
#endif
		if( (pParse->db->flags & SQLITE_CountRows)!=0 ){
			return 0;  /* xfer opt does not play well with PRAGMA count_changes */
		}

		/* If we get this far, it means that the xfer optimization is at
		** least a possibility, though it might only work if the destination
		** table (tab1) is initially empty.
		*/
#ifdef TEST
		sqlite3_xferopt_count++;
#endif
		iDbSrc = sqlite3SchemaToIndex(pParse->db, pSrc->pSchema);
		v = sqlite3GetVdbe(pParse);
		sqlite3CodeVerifySchema(pParse, iDbSrc);
		iSrc = pParse->nTab++;
		iDest = pParse->nTab++;
		regAutoinc = autoIncBegin(pParse, iDbDest, pDest);
		sqlite3OpenTable(pParse, iDest, iDbDest, pDest, OP_OpenWrite);
		if( (pDest->iPKey<0 && pDest->pIndex!=0)          /* (1) */
			|| destHasUniqueIdx                              /* (2) */
			|| (onError!=OE_Abort && onError!=OE_Rollback)   /* (3) */
			){
				/* In some circumstances, we are able to run the xfer optimization
				** only if the destination table is initially empty.  This code makes
				** that determination.  Conditions under which the destination must
				** be empty:
				**
				** (1) There is no INTEGER PRIMARY KEY but there are indices.
				**     (If the destination is not initially empty, the rowid fields
				**     of index entries might need to change.)
				**
				** (2) The destination has a unique index.  (The xfer optimization 
				**     is unable to test uniqueness.)
				**
				** (3) onError is something other than OE_Abort and OE_Rollback.
				*/
				addr1 = sqlite3VdbeAddOp2(v, OP_Rewind, iDest, 0);
				emptyDestTest = sqlite3VdbeAddOp2(v, OP_Goto, 0, 0);
				sqlite3VdbeJumpHere(v, addr1);
		}else{
			emptyDestTest = 0;
		}
		sqlite3OpenTable(pParse, iSrc, iDbSrc, pSrc, OP_OpenRead);
		emptySrcTest = sqlite3VdbeAddOp2(v, OP_Rewind, iSrc, 0);
		regData = sqlite3GetTempReg(pParse);
		regRowid = sqlite3GetTempReg(pParse);
		if( pDest->iPKey>=0 ){
			addr1 = sqlite3VdbeAddOp2(v, OP_Rowid, iSrc, regRowid);
			addr2 = sqlite3VdbeAddOp3(v, OP_NotExists, iDest, 0, regRowid);
			sqlite3HaltConstraint(pParse, SQLITE_CONSTRAINT_PRIMARYKEY,
				onError, "PRIMARY KEY must be unique", P4_STATIC);
			sqlite3VdbeJumpHere(v, addr2);
			autoIncStep(pParse, regAutoinc, regRowid);
		}else if( pDest->pIndex==0 ){
			addr1 = sqlite3VdbeAddOp2(v, OP_NewRowid, iDest, regRowid);
		}else{
			addr1 = sqlite3VdbeAddOp2(v, OP_Rowid, iSrc, regRowid);
			assert( (pDest->tabFlags & TF_Autoincrement)==0 );
		}
		sqlite3VdbeAddOp2(v, OP_RowData, iSrc, regData);
		sqlite3VdbeAddOp3(v, OP_Insert, iDest, regData, regRowid);
		sqlite3VdbeChangeP5(v, OPFLAG_NCHANGE|OPFLAG_LASTROWID|OPFLAG_APPEND);
		sqlite3VdbeChangeP4(v, -1, pDest->zName, 0);
		sqlite3VdbeAddOp2(v, OP_Next, iSrc, addr1);
		for(pDestIdx=pDest->pIndex; pDestIdx; pDestIdx=pDestIdx->pNext){
			for(pSrcIdx=pSrc->pIndex; ALWAYS(pSrcIdx); pSrcIdx=pSrcIdx->pNext){
				if( xferCompatibleIndex(pDestIdx, pSrcIdx) ) break;
			}
			assert( pSrcIdx );
			sqlite3VdbeAddOp2(v, OP_Close, iSrc, 0);
			sqlite3VdbeAddOp2(v, OP_Close, iDest, 0);
			pKey = sqlite3IndexKeyinfo(pParse, pSrcIdx);
			sqlite3VdbeAddOp4(v, OP_OpenRead, iSrc, pSrcIdx->tnum, iDbSrc,
				(char*)pKey, P4_KEYINFO_HANDOFF);
			VdbeComment((v, "%s", pSrcIdx->zName));
			pKey = sqlite3IndexKeyinfo(pParse, pDestIdx);
			sqlite3VdbeAddOp4(v, OP_OpenWrite, iDest, pDestIdx->tnum, iDbDest,
				(char*)pKey, P4_KEYINFO_HANDOFF);
			VdbeComment((v, "%s", pDestIdx->zName));
			addr1 = sqlite3VdbeAddOp2(v, OP_Rewind, iSrc, 0);
			sqlite3VdbeAddOp2(v, OP_RowKey, iSrc, regData);
			sqlite3VdbeAddOp3(v, OP_IdxInsert, iDest, regData, 1);
			sqlite3VdbeAddOp2(v, OP_Next, iSrc, addr1+1);
			sqlite3VdbeJumpHere(v, addr1);
		}
		sqlite3VdbeJumpHere(v, emptySrcTest);
		sqlite3ReleaseTempReg(pParse, regRowid);
		sqlite3ReleaseTempReg(pParse, regData);
		sqlite3VdbeAddOp2(v, OP_Close, iSrc, 0);
		sqlite3VdbeAddOp2(v, OP_Close, iDest, 0);
		if( emptyDestTest ){
			sqlite3VdbeAddOp2(v, OP_Halt, SQLITE_OK, 0);
			sqlite3VdbeJumpHere(v, emptyDestTest);
			sqlite3VdbeAddOp2(v, OP_Close, iDest, 0);
			return 0;
		}else{
			return 1;
		}
	}
#endif

}}