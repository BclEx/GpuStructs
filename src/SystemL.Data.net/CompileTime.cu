#pragma region OMIT_COMPILEOPTION_DIAGS
#ifndef OMIT_COMPILEOPTION_DIAGS
#include <Core/Core.cu.h>

namespace Core
{
	// These macros are provided to "stringify" the value of the define for those options in which the value is meaningful.
#define CTIMEOPT_VAL_(opt) #opt
#define CTIMEOPT_VAL(opt) CTIMEOPT_VAL_(opt)

	__constant__ static const char *const _compileOpt[] = {
#ifdef _32BIT_ROWID
		"32BIT_ROWID",
#endif
#ifdef _4_BYTE_ALIGNED_MALLOC
		"4_BYTE_ALIGNED_MALLOC",
#endif
#ifdef CASE_SENSITIVE_LIKE
		"CASE_SENSITIVE_LIKE",
#endif
#ifdef CHECK_PAGES
		"CHECK_PAGES",
#endif
#ifdef COVERAGE_TEST
		"COVERAGE_TEST",
#endif
#ifdef CURDIR
		"CURDIR",
#endif
#ifdef _DEBUG
		"DEBUG",
#endif
#ifdef DEFAULT_LOCKING_MODE
		"DEFAULT_LOCKING_MODE=" CTIMEOPT_VAL(DEFAULT_LOCKING_MODE),
#endif
#ifdef DISABLE_DIRSYNC
		"DISABLE_DIRSYNC",
#endif
#ifdef DISABLE_LFS
		"DISABLE_LFS",
#endif
#ifdef ENABLE_ATOMIC_WRITE
		"ENABLE_ATOMIC_WRITE",
#endif
#ifdef ENABLE_CEROD
		"ENABLE_CEROD",
#endif
#ifdef ENABLE_COLUMN_METADATA
		"ENABLE_COLUMN_METADATA",
#endif
#ifdef ENABLE_EXPENSIVE_ASSERT
		"ENABLE_EXPENSIVE_ASSERT",
#endif
#ifdef ENABLE_FTS1
		"ENABLE_FTS1",
#endif
#ifdef ENABLE_FTS2
		"ENABLE_FTS2",
#endif
#ifdef ENABLE_FTS3
		"ENABLE_FTS3",
#endif
#ifdef ENABLE_FTS3_PARENTHESIS
		"ENABLE_FTS3_PARENTHESIS",
#endif
#ifdef ENABLE_FTS4
		"ENABLE_FTS4",
#endif
#ifdef ENABLE_ICU
		"ENABLE_ICU",
#endif
#ifdef ENABLE_IOTRACE
		"ENABLE_IOTRACE",
#endif
#ifdef ENABLE_LOAD_EXTENSION
		"ENABLE_LOAD_EXTENSION",
#endif
#ifdef ENABLE_LOCKING_STYLE
		"ENABLE_LOCKING_STYLE=" CTIMEOPT_VAL(ENABLE_LOCKING_STYLE),
#endif
#ifdef ENABLE_MEMORY_MANAGEMENT
		"ENABLE_MEMORY_MANAGEMENT",
#endif
#ifdef ENABLE_MEMSYS3
		"ENABLE_MEMSYS3",
#endif
#ifdef ENABLE_MEMSYS5
		"ENABLE_MEMSYS5",
#endif
#ifdef ENABLE_OVERSIZE_CELL_CHECK
		"ENABLE_OVERSIZE_CELL_CHECK",
#endif
#ifdef ENABLE_RTREE
		"ENABLE_RTREE",
#endif
#ifdef ENABLE_STAT3
		"ENABLE_STAT3",
#endif
#ifdef ENABLE_UNLOCK_NOTIFY
		"ENABLE_UNLOCK_NOTIFY",
#endif
#ifdef ENABLE_UPDATE_DELETE_LIMIT
		"ENABLE_UPDATE_DELETE_LIMIT",
#endif
#ifdef HAS_CODEC
		"HAS_CODEC",
#endif
#ifdef HAVE_ISNAN
		"HAVE_ISNAN",
#endif
#ifdef HOMEGROWN_RECURSIVE_MUTEX
		"HOMEGROWN_RECURSIVE_MUTEX",
#endif
#ifdef IGNORE_AFP_LOCK_ERRORS
		"IGNORE_AFP_LOCK_ERRORS",
#endif
#ifdef IGNORE_FLOCK_LOCK_ERRORS
		"IGNORE_FLOCK_LOCK_ERRORS",
#endif
#ifdef INT64_TYPE
		"INT64_TYPE",
#endif
#ifdef LOCK_TRACE
		"LOCK_TRACE",
#endif
#ifdef MAX_SCHEMA_RETRY
		"MAX_SCHEMA_RETRY=" CTIMEOPT_VAL(MAX_SCHEMA_RETRY),
#endif
#ifdef MEMDEBUG
		"MEMDEBUG",
#endif
#ifdef MIXED_ENDIAN_64BIT_FLOAT
		"MIXED_ENDIAN_64BIT_FLOAT",
#endif
#ifdef NO_SYNC
		"NO_SYNC",
#endif
#ifdef OMIT_ALTERTABLE
		"OMIT_ALTERTABLE",
#endif
#ifdef OMIT_ANALYZE
		"OMIT_ANALYZE",
#endif
#ifdef OMIT_ATTACH
		"OMIT_ATTACH",
#endif
#ifdef OMIT_AUTHORIZATION
		"OMIT_AUTHORIZATION",
#endif
#ifdef OMIT_AUTOINCREMENT
		"OMIT_AUTOINCREMENT",
#endif
#ifdef OMIT_AUTOINIT
		"OMIT_AUTOINIT",
#endif
#ifdef OMIT_AUTOMATIC_INDEX
		"OMIT_AUTOMATIC_INDEX",
#endif
#ifdef OMIT_AUTORESET
		"OMIT_AUTORESET",
#endif
#ifdef OMIT_AUTOVACUUM
		"OMIT_AUTOVACUUM",
#endif
#ifdef OMIT_BETWEEN_OPTIMIZATION
		"OMIT_BETWEEN_OPTIMIZATION",
#endif
#ifdef OMIT_BLOB_LITERAL
		"OMIT_BLOB_LITERAL",
#endif
#ifdef OMIT_BTREECOUNT
		"OMIT_BTREECOUNT",
#endif
#ifdef OMIT_BUILTIN_TEST
		"OMIT_BUILTIN_TEST",
#endif
#ifdef OMIT_CAST
		"OMIT_CAST",
#endif
#ifdef OMIT_CHECK
		"OMIT_CHECK",
#endif
		// redundant
		//#ifdef OMIT_COMPILEOPTION_DIAGS
		//   "OMIT_COMPILEOPTION_DIAGS",
		//#endif
#ifdef OMIT_COMPLETE
		"OMIT_COMPLETE",
#endif
#ifdef OMIT_COMPOUND_SELECT
		"OMIT_COMPOUND_SELECT",
#endif
#ifdef OMIT_DATETIME_FUNCS
		"OMIT_DATETIME_FUNCS",
#endif
#ifdef OMIT_DECLTYPE
		"OMIT_DECLTYPE",
#endif
#ifdef OMIT_DEPRECATED
		"OMIT_DEPRECATED",
#endif
#ifdef OMIT_DISKIO
		"OMIT_DISKIO",
#endif
#ifdef OMIT_EXPLAIN
		"OMIT_EXPLAIN",
#endif
#ifdef OMIT_FLAG_PRAGMAS
		"OMIT_FLAG_PRAGMAS",
#endif
#ifdef OMIT_FLOATING_POINT
		"OMIT_FLOATING_POINT",
#endif
#ifdef OMIT_FOREIGN_KEY
		"OMIT_FOREIGN_KEY",
#endif
#ifdef OMIT_GET_TABLE
		"OMIT_GET_TABLE",
#endif
#ifdef OMIT_INCRBLOB
		"OMIT_INCRBLOB",
#endif
#ifdef OMIT_INTEGRITY_CHECK
		"OMIT_INTEGRITY_CHECK",
#endif
#ifdef OMIT_LIKE_OPTIMIZATION
		"OMIT_LIKE_OPTIMIZATION",
#endif
#ifdef OMIT_LOAD_EXTENSION
		"OMIT_LOAD_EXTENSION",
#endif
#ifdef OMIT_LOCALTIME
		"OMIT_LOCALTIME",
#endif
#ifdef OMIT_LOOKASIDE
		"OMIT_LOOKASIDE",
#endif
#ifdef OMIT_MEMORYDB
		"OMIT_MEMORYDB",
#endif
#ifdef OMIT_OR_OPTIMIZATION
		"OMIT_OR_OPTIMIZATION",
#endif
#ifdef OMIT_PAGER_PRAGMAS
		"OMIT_PAGER_PRAGMAS",
#endif
#ifdef OMIT_PRAGMA
		"OMIT_PRAGMA",
#endif
#ifdef OMIT_PROGRESS_CALLBACK
		"OMIT_PROGRESS_CALLBACK",
#endif
#ifdef OMIT_QUICKBALANCE
		"OMIT_QUICKBALANCE",
#endif
#ifdef OMIT_REINDEX
		"OMIT_REINDEX",
#endif
#ifdef OMIT_SCHEMA_PRAGMAS
		"OMIT_SCHEMA_PRAGMAS",
#endif
#ifdef OMIT_SCHEMA_VERSION_PRAGMAS
		"OMIT_SCHEMA_VERSION_PRAGMAS",
#endif
#ifdef OMIT_SHARED_CACHE
		"OMIT_SHARED_CACHE",
#endif
#ifdef OMIT_SUBQUERY
		"OMIT_SUBQUERY",
#endif
#ifdef OMIT_TCL_VARIABLE
		"OMIT_TCL_VARIABLE",
#endif
#ifdef OMIT_TEMPDB
		"OMIT_TEMPDB",
#endif
#ifdef OMIT_TRACE
		"OMIT_TRACE",
#endif
#ifdef OMIT_TRIGGER
		"OMIT_TRIGGER",
#endif
#ifdef OMIT_TRUNCATE_OPTIMIZATION
		"OMIT_TRUNCATE_OPTIMIZATION",
#endif
#ifdef OMIT_UTF16
		"OMIT_UTF16",
#endif
#ifdef OMIT_VACUUM
		"OMIT_VACUUM",
#endif
#ifdef OMIT_VIEW
		"OMIT_VIEW",
#endif
#ifdef OMIT_VIRTUALTABLE
		"OMIT_VIRTUALTABLE",
#endif
#ifdef OMIT_WAL
		"OMIT_WAL",
#endif
#ifdef OMIT_WSD
		"OMIT_WSD",
#endif
#ifdef OMIT_XFER_OPT
		"OMIT_XFER_OPT",
#endif
#ifdef PERFORMANCE_TRACE
		"PERFORMANCE_TRACE",
#endif
#ifdef PROXY_DEBUG
		"PROXY_DEBUG",
#endif
#ifdef RTREE_INT_ONLY
		"RTREE_INT_ONLY",
#endif
#ifdef SECURE_DELETE
		"SECURE_DELETE",
#endif
#ifdef SMALL_STACK
		"SMALL_STACK",
#endif
#ifdef SOUNDEX
		"SOUNDEX",
#endif
#ifdef TCL
		"TCL",
#endif
#ifdef TEMP_STORE
		"TEMP_STORE=" CTIMEOPT_VAL(TEMP_STORE),
#endif
#ifdef TEST
		"TEST",
#endif
#ifdef THREADSAFE
		"THREADSAFE=" CTIMEOPT_VAL(THREADSAFE),
#endif
#ifdef USE_ALLOCA
		"USE_ALLOCA",
#endif
#ifdef ZERO_MALLOC
		"ZERO_MALLOC"
#endif
	};

	__device__ bool CompileTimeOptionUsed(const char *optName)
	{
		if (!_strncmp(optName, "", 7)) optName += 7;
		int length = _strlen30(optName);
		// Since ArraySize(azCompileOpt) is normally in single digits, a linear search is adequate.  No need for a binary search.
		for (int i = 0; i < _lengthof(_compileOpt); i++)
			if (!_strncmp(optName, _compileOpt[i], length) && (_compileOpt[i][length] == 0 || _compileOpt[i][length] == '='))
				return true;
		return false;
	}

	__device__ const char *CompileTimeGet(int id)
	{
		return (id >= 0 && id < _lengthof(_compileOpt) ? _compileOpt[id] : nullptr);
	}

}
#endif
#pragma endregion