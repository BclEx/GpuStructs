#include "hip/hip_runtime.h"
﻿// os.c
#include "Core.cu.h"
using namespace Core::IO;
namespace Core
{
	__device__ static VSystem *_WSD g_vfsList = nullptr;
#define _vfsList _GLOBAL(VSystem *, g_vfsList)

	__device__ VSystem *VSystem::FindVfs(const char *name)
	{
#ifndef OMIT_AUTOINIT
		RC rc = SysEx::AutoInitialize();
		if (rc) return nullptr;
#endif
		VSystem *vfs = nullptr;
		MutexEx mutex = MutexEx::Alloc(MutexEx::MUTEX_STATIC_MASTER);
		MutexEx::Enter(mutex);
		for (vfs = _vfsList; vfs && _strcmp(name, vfs->Name); vfs = vfs->Next) { }
		MutexEx::Leave(mutex);
		return vfs;
	}

	__device__ static void UnlinkVfs(VSystem *vfs)
	{
		_assert(MutexEx::Held(MutexEx::Alloc(MutexEx::MUTEX_STATIC_MASTER)));
		if (!vfs) { }
		else if (_vfsList == vfs)
			_vfsList = vfs->Next;
		else if (_vfsList)
		{
			VSystem *p = _vfsList;
			while (p->Next && p->Next != vfs)
				p = p->Next;
			if (p->Next == vfs)
				p->Next = vfs->Next;
		}
	}

	__device__ int VSystem::RegisterVfs(VSystem *vfs, bool default_)
	{
		MutexEx mutex = MutexEx::Alloc(MutexEx::MUTEX_STATIC_MASTER);
		MutexEx::Enter(mutex);
		UnlinkVfs(vfs);
		if (default_ || !_vfsList)
		{
			vfs->Next = _vfsList;
			_vfsList = vfs;
		}
		else
		{
			vfs->Next = _vfsList->Next;
			_vfsList->Next = vfs;
		}
		_assert(_vfsList != nullptr);
		MutexEx::Leave(mutex);
		return RC_OK;
	}

	__device__ int VSystem::UnregisterVfs(VSystem *vfs)
	{
		MutexEx mutex = MutexEx::Alloc(MutexEx::MUTEX_STATIC_MASTER);
		MutexEx::Enter(mutex);
		UnlinkVfs(vfs);
		MutexEx::Leave(mutex);
		return RC_OK;
	}

	// from main_c
#pragma region File

#ifdef ENABLE_8_3_NAMES
	__device__ void SysEx::FileSuffix3(const char *baseFilename, char *z)
	{
#if ENABLE_8_3_NAMES<2
		if (!UriBoolean(baseFilename, "8_3_names", 0)) return;
#endif
		int size = _strlen30(z);
		int i;
		for (i = size-1; i > 0 && z[i] != '/' && z[i] !='.'; i--) { }
		if (z[i] == '.' && _ALWAYS(size > i+4)) _memmove(&z[i+1], &z[size-3], 4);
	}
#endif

	struct OpenMode
	{
		const char *Z;
		VSystem::OPEN Mode;
	};


	//SKY TODO: transfer host to constant
//#if __HIPCC__
//	__constant__ static OpenMode _cacheModes[3];
//	__constant__ static OpenMode h_cacheModes[3] =
//#else
//	static OpenMode _cacheModes[] =
//#endif
	__constant__ static OpenMode _cacheModes[3] =
	{
		{ "shared",  VSystem::OPEN_SHAREDCACHE },
		{ "private", VSystem::OPEN_PRIVATECACHE },
		{ nullptr, (VSystem::OPEN)0 }
	};

#if __HIPCC__
	__constant__ static OpenMode _openModes[5];
	static OpenMode h_openModes[] =
#else
	static OpenMode _openModes[] =
#endif
	//__constant__ static OpenMode _openModes[5] =
	{
		{ "ro",  VSystem::OPEN_READONLY },
		{ "rw",  VSystem::OPEN_READWRITE }, 
		{ "rwc", VSystem::OPEN_READWRITE | VSystem::OPEN_CREATE },
		{ "memory", VSystem::OPEN_MEMORY },
		{ nullptr, (VSystem::OPEN)0 }
	};

	__device__ RC VSystem::ParseUri(const char *defaultVfsName, const char *uri, VSystem::OPEN *flagsRef, VSystem **vfsOut, char **fileNameOut, char **errMsgOut)
	{
		_assert(*errMsgOut == nullptr);

		VSystem::OPEN flags = *flagsRef;
		const char *vfsName = defaultVfsName;
		int uriLength = _strlen30(uri);

		RC rc = RC_OK;
		char *fileName;
		if (((flags & VSystem::OPEN_URI) || SysEx_GlobalStatics.OpenUri) && uriLength >= 5 && !_memcmp(uri, "file:", 5))
		{
			// Make sure the SQLITE_OPEN_URI flag is set to indicate to the VFS xOpen method that there may be extra parameters following the file-name.
			flags |= VSystem::OPEN_URI;

			int bytes = uriLength+2; // Bytes of space to allocate
			int uriIdx; // Input character index
			for (uriIdx = 0; uriIdx < uriLength; uriIdx++) bytes += (uri[uriIdx] == '&');
			fileName = (char *)_alloc(bytes);
			if (!fileName) return RC_NOMEM;

			// Discard the scheme and authority segments of the URI.
			if (uri[5] == '/' && uri[6] == '/')
			{
				uriIdx = 7;
				while (uri[uriIdx] && uri[uriIdx] != '/') uriIdx++;
				if (uriIdx != 7 && (uriIdx != 16 || _memcmp("localhost", &uri[7], 9)))
				{
					*errMsgOut = _mprintf("invalid uri authority: %.*s", uriIdx-7, &uri[7]);
					rc = RC_ERROR;
					goto parse_uri_out;
				}
			}
			else
				uriIdx = 5;

			// Copy the filename and any query parameters into the zFile buffer. Decode %HH escape codes along the way. 
			//
			// Within this loop, variable eState may be set to 0, 1 or 2, depending on the parsing context. As follows:
			//
			//   0: Parsing file-name.
			//   1: Parsing name section of a name=value query parameter.
			//   2: Parsing value section of a name=value query parameter.
			int state = 0; // Parser state when parsing URI
			char c;
			int fileNameIdx = 0; // Output character index
			while ((c = uri[uriIdx]) != 0 && c != '#')
			{
				uriIdx++;
				if (c == '%' && _isxdigit(uri[uriIdx]) && _isxdigit(uri[uriIdx+1]))
				{
					int octet = (_hextobyte(uri[uriIdx++]) << 4);
					octet += _hextobyte(uri[uriIdx++]);
					_assert(octet >= 0 && octet < 256);
					if (octet == 0)
					{
						// This branch is taken when "%00" appears within the URI. In this case we ignore all text in the remainder of the path, name or
						// value currently being parsed. So ignore the current character and skip to the next "?", "=" or "&", as appropriate.
						while ((c = uri[uriIdx]) != 0 && c !='#' && 
							(state != 0 || c != '?') && 
							(state != 1 || (c != '=' && c != '&')) && 
							(state != 2 || c != '&'))
							uriIdx++;
						continue;
					}
					c = octet;
				}
				else if (state == 1 && (c == '&' || c == '='))
				{
					if (fileName[fileNameIdx-1] == 0)
					{
						// An empty option name. Ignore this option altogether.
						while (uri[uriIdx] && uri[uriIdx] != '#' && uri[uriIdx-1] != '&') uriIdx++;
						continue;
					}
					if (c == '&')
						fileName[fileNameIdx++] = '\0';
					else
						state = 2;
					c = 0;
				}
				else if ((state == 0 && c == '?') || (state == 2 && c == '&'))
				{
					c = 0;
					state = 1;
				}
				fileName[fileNameIdx++] = c;
			}
			if (state == 1) fileName[fileNameIdx++] = '\0';
			fileName[fileNameIdx++] = '\0';
			fileName[fileNameIdx++] = '\0';

			// Check if there were any options specified that should be interpreted here. Options that are interpreted here include "vfs" and those that
			// correspond to flags that may be passed to the sqlite3_open_v2() method.
			char *opt = &fileName[_strlen30(fileName)+1];
			while (opt[0])
			{
				int optLength = _strlen30(opt);
				char *val = &opt[optLength+1];
				int valLength = _strlen30(val);
				if (optLength == 3 && !_memcmp("vfs", opt, 3))
					vfsName = val;
				else
				{
					OpenMode *modes = nullptr;
					char *modeType = nullptr;
					VSystem::OPEN mask = (VSystem::OPEN)0;
					VSystem::OPEN limit = (VSystem::OPEN)0;
					if (optLength == 5 && !_memcmp("cache", opt, 5))
					{
						mask = VSystem::OPEN_SHAREDCACHE | VSystem::OPEN_PRIVATECACHE;
						modes = _cacheModes;
						limit = mask;
						modeType = "cache";
					}
					if (optLength == 4 && !_memcmp("mode", opt, 4))
					{
						mask = VSystem::OPEN_READONLY | VSystem::OPEN_READWRITE | VSystem::OPEN_CREATE | VSystem::OPEN_MEMORY;
						modes = _openModes;
						limit = mask & flags;
						modeType = "access";
					}
					if (modes)
					{
						VSystem::OPEN mode = (VSystem::OPEN)0;
						for (int i = 0; modes[i].Z; i++)
						{
							const char *z = modes[i].Z;
							if (valLength == _strlen30(z) && !_memcmp(val, z, valLength))
							{
								mode = modes[i].Mode;
								break;
							}
						}
						if (mode == 0)
						{
							*errMsgOut = _mprintf("no such %s mode: %s", modeType, val);
							rc = RC_ERROR;
							goto parse_uri_out;
						}
						if ((mode & ~VSystem::OPEN_MEMORY) > limit)
						{
							*errMsgOut = _mprintf("%s mode not allowed: %s", modeType, val);
							rc = RC_PERM;
							goto parse_uri_out;
						}
						flags = (VSystem::OPEN)(flags & ~mask) | mode;
					}
				}
				opt = &val[valLength+1];
			}
		}
		else
		{
			fileName = (char *)_alloc(uriLength+2);
			if (!fileName) return RC_NOMEM;
			_memcpy(fileName, uri, uriLength);
			fileName[uriLength] = '\0';
			fileName[uriLength+1] = '\0';
			flags &= ~VSystem::OPEN_URI;
		}

		*vfsOut = FindVfs(vfsName);
		if (!*vfsOut)
		{
			*errMsgOut = _mprintf("no such vfs: %s", vfsName);
			rc = RC_ERROR;
		}

parse_uri_out:
		if (rc != RC_OK)
		{
			_free(fileName);
			fileName = nullptr;
		}
		*flagsRef = flags;
		*fileNameOut = fileName;
		return rc;
	}

	__device__ const char *VSystem::UriParameter(const char *filename, const char *param)
	{
		if (!filename) return nullptr;
		filename += _strlen30(filename) + 1;
		while (filename[0])
		{
			int x = _strcmp(filename, param);
			filename += _strlen30(filename) + 1;
			if (x == 0) return filename;
			filename += _strlen30(filename) + 1;
		}
		return nullptr;
	}

	__device__ bool VSystem::UriBoolean(const char *filename, const char *param, bool dflt)
	{
		const char *z = UriParameter(filename, param);
		return (z ? ConvertEx::GetBoolean(z, dflt) : dflt);
	}

	__device__ int64 VSystem::UriInt64(const char *filename, const char *param, int64 dflt)
	{
		const char *z = UriParameter(filename, param);
		int64 v;
		return (z && ConvertEx::Atoi64(z, &v, _strlen30(z), TEXTENCODE_UTF8) == RC_OK ? v : dflt);
	}

#pragma endregion
}