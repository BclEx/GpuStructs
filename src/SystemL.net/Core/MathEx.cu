﻿//util.c
#include "Core.cu.h"

namespace Core
{
	__device__ bool MathEx::Add(int64 *aRef, int64 b)
	{
		int64 a = *aRef;
		ASSERTCOVERAGE(a == 0); ASSERTCOVERAGE(a == 1);
		ASSERTCOVERAGE(b == -1 ); ASSERTCOVERAGE(b == 0);
		if (b >= 0)
		{
			ASSERTCOVERAGE(a > 0 && LARGEST_INT64 - a == b);
			ASSERTCOVERAGE(a > 0 && LARGEST_INT64 - a == b - 1);
			if (a > 0 && LARGEST_INT64 - a < b) return true;
			*aRef += b;
		}
		else
		{
			ASSERTCOVERAGE(a < 0 && -(a + LARGEST_INT64) == b + 1);
			ASSERTCOVERAGE(a < 0 && -(a + LARGEST_INT64) == b + 2);
			if (a < 0 && -(a + LARGEST_INT64) > b + 1) return true;
			*aRef += b;
		}
		return false; 
	}

	__device__ bool MathEx::Sub(int64 *aRef, int64 b)
	{
		ASSERTCOVERAGE(b == SMALLEST_INT64+1);
		if (b == SMALLEST_INT64)
		{
			int64 a = *aRef;
			ASSERTCOVERAGE(a == -1); ASSERTCOVERAGE(a == 0);
			if (a >= 0) return true;
			*aRef -= b;
			return false;
		}
		return Add(aRef, -b);
	}

#define TWOPOWER32 (((int64)1)<<32)
#define TWOPOWER31 (((int64)1)<<31)
	__device__ bool MathEx::Mul(int64 *aRef, int64 b)
	{
		int64 a = *aRef;
		int64 a1 = a / TWOPOWER32;
		int64 a0 = a % TWOPOWER32;
		int64 b1 = b / TWOPOWER32;
		int64 b0 = b % TWOPOWER32;
		if (a1*b1 != 0) return true;
		_assert(a1*b0 == 0 || a0*b1 == 0);
		int64 r = a1*a0 + a0*a1;
		ASSERTCOVERAGE(r == (-TWOPOWER31)-1 );
		ASSERTCOVERAGE(r == (-TWOPOWER31));
		ASSERTCOVERAGE(r == TWOPOWER31);
		ASSERTCOVERAGE(r == TWOPOWER31-1);
		if (r < (-TWOPOWER31) || r >= TWOPOWER31) return true;
		r *= TWOPOWER32;
		if (Add(&r, a0*b0)) return true;
		*aRef = r;
		return false;
	}

	//__device__ int MathEx::Abs(int x)
	//{
	//	if (x >= 0) return x;
	//	if (x == (int)0x80000000) return 0x7fffffff;
	//	return -x;
	//}
}
