#include "hip/hip_runtime.h"
﻿#include "../Core.cu.h"

namespace Core { namespace Text
{

#pragma region Printf

#ifndef PRINT_BUF_SIZE
#define PRINT_BUF_SIZE 70
#endif
#define BUFSIZE PRINT_BUF_SIZE  // Size of the output buffer

	enum TYPE : uint8
	{
		TYPE_RADIX = 1, // Integer types.  %d, %x, %o, and so forth
		TYPE_TYPE_FLOAT = 2, // Floating point.  %f
		TYPE_TYPE_EXP = 3, // Exponentional notation. %e and %E
		TYPE_TYPE_GENERIC = 4, // Floating or exponential, depending on exponent. %g
		TYPE_SIZE = 5, // Return number of characters processed so far. %n
		TYPE_STRING = 6, // Strings. %s
		TYPE_DYNSTRING = 7, // Dynamically allocated strings. %z
		TYPE_PERCENT = 8, // Percent symbol. %%
		TYPE_CHARX = 9, // Characters. %c
		// The rest are extensions, not normally found in printf()
		TYPE_SQLESCAPE = 10, // Strings with '\'' doubled.  %q
		TYPE_SQLESCAPE2 = 11, // Strings with '\'' doubled and enclosed in '', NULL pointers replaced by SQL NULL.  %Q
		TYPE_TOKEN = 12, // a pointer to a Token structure
		TYPE_SRCLIST = 13, // a pointer to a SrcList
		TYPE_POINTER = 14, // The %p conversion
		TYPE_SQLESCAPE3 = 15, // %w -> Strings with '\"' doubled
		TYPE_ORDINAL = 16, // %r -> 1st, 2nd, 3rd, 4th, etc.  English only
		//
		TYPE_INVALID = 0, // Any unrecognized conversion type
	};

	enum FLAG : uint8
	{
		FLAG_SIGNED = 1,     // True if the value to convert is signed
		FLAG_INTERN = 2,     // True if for internal use only
		FLAG_STRING = 4,     // Allow infinity precision
	};

	// Each builtin conversion character (ex: the 'd' in "%d") is described by an instance of the following structure
	struct Info
	{   // Information about each format field
		char Fmttype; // The format field code letter
		uint8 Base; // The base for radix conversion
		FLAG Flags; // One or more of FLAG_ constants below
		TYPE Type; // Conversion paradigm
		uint8 Charset; // Offset into aDigits[] of the digits string
		uint8 Prefix; // Offset into aPrefix[] of the prefix string
	};

	// The following table is searched linearly, so it is good to put the most frequently used conversion types first.
	__device__ static const char _digits[] = "0123456789ABCDEF0123456789abcdef";
	__device__ static const char _prefix[] = "-x0\000X0";
	__device__ static const Info _info[] = {
		{ 'd', 10, (FLAG)1, TYPE_RADIX,      0,  0 },
		{ 's',  0, (FLAG)4, TYPE_STRING,     0,  0 },
		{ 'g',  0, (FLAG)1, TYPE_GENERIC,    30, 0 },
		{ 'z',  0, (FLAG)4, TYPE_DYNSTRING,  0,  0 },
		{ 'q',  0, (FLAG)4, TYPE_SQLESCAPE,  0,  0 },
		{ 'Q',  0, (FLAG)4, TYPE_SQLESCAPE2, 0,  0 },
		{ 'w',  0, (FLAG)4, TYPE_SQLESCAPE3, 0,  0 },
		{ 'c',  0, (FLAG)0, TYPE_CHARX,      0,  0 },
		{ 'o',  8, (FLAG)0, TYPE_RADIX,      0,  2 },
		{ 'u', 10, (FLAG)0, TYPE_RADIX,      0,  0 },
		{ 'x', 16, (FLAG)0, TYPE_RADIX,      16, 1 },
		{ 'X', 16, (FLAG)0, TYPE_RADIX,      0,  4 },
#ifndef OMIT_FLOATING_POINT
		{ 'f',  0, (FLAG)1, TYPE_FLOAT,      0,  0 },
		{ 'e',  0, (FLAG)1, TYPE_EXP,        30, 0 },
		{ 'E',  0, (FLAG)1, TYPE_EXP,        14, 0 },
		{ 'G',  0, (FLAG)1, TYPE_GENERIC,    14, 0 },
#endif
		{ 'i', 10, (FLAG)1, TYPE_RADIX,      0,  0 },
		{ 'n',  0, (FLAG)0, TYPE_SIZE,       0,  0 },
		{ '%',  0, (FLAG)0, TYPE_PERCENT,    0,  0 },
		{ 'p', 16, (FLAG)0, TYPE_POINTER,    0,  1 },
		// All the rest have the FLAG_INTERN bit set and are thus for internal use only
		{ 'T',  0, (FLAG)2, TYPE_TOKEN,      0,  0 },
		{ 'S',  0, (FLAG)2, TYPE_SRCLIST,    0,  0 },
		{ 'r', 10, (FLAG)3, TYPE_ORDINAL,    0,  0 },
	};

#ifndef OMIT_FLOATING_POINT
	__device__ static char GetDigit(double64 *val, int *cnt)
	{
		if ((*cnt) <= 0) return '0';
		(*cnt)--;
		int digit = (int)*val;
		double64 d = digit;
		digit += '0';
		*val = (*val - d)*10.0;
		return (char)digit;
	}
#endif

	__constant__ static const char _spaces[] = "                             ";
	__device__ static void AppendSpace(StringBuilder *b, int length)
	{
		while (length >= (int)sizeof(_spaces)-1)
		{
			b->Append(_spaces, sizeof(_spaces)-1);
			length -= sizeof(_spaces)-1;
		}
		if (length > 0)
			b->Append, _spaces, length);
	}

	__constant__ static const char _ord[] = "thstndrd";
	void Printf(StringBuilder *b, bool useExtended, const char *fmt, _argsT args) //+ sqlite3VXPrintf
	{
		char buf[BUFSIZE]; // Conversion buffer
		char *bufpt = nullptr; // Pointer to the conversion buffer
		int c; // Next character in the format string
		bool flag_leftjustify = false; // True if "-" flag is present
		int width = 0; // Width of the current field
		int length = 0; // Length of the field
		for (; (c = (*fmt)) != 0; ++fmt)
		{
			if (c != '%')
			{
				bufpt = (char *)fmt;
				int amt = 1;
				while ((c = (*++fmt)) != '%' && c != 0) amt++;
				b->Append(bufpt, amt);
				if (c == 0) break;
			}
			if ((c = (*++fmt)) == 0)
			{
				b->Append("%", 1);
				break;
			}
			// Find out what flags are present
			flag_leftjustify = false; // True if "-" flag is present
			bool flag_plussign = false; // True if "+" flag is present
			bool flag_blanksign = false; // True if " " flag is present
			bool flag_alternateform = false; // True if "#" flag is present
			bool flag_altform2 = false; // True if "!" flag is present
			bool flag_zeropad = false; // True if field width constant starts with zero
			bool done = false; // Loop termination flag
			do
			{
				switch (c)
				{
				case '-': flag_leftjustify = true; break;
				case '+': flag_plussign = true; break;
				case ' ': flag_blanksign = true; break;
				case '#': flag_alternateform = true; break;
				case '!': flag_altform2 = true; break;
				case '0': flag_zeropad = true; break;
				default: done = true; break;
				}
			} while (!done && (c = (*++fmt)) != 0);
			// Get the field width
			width = 0; // Width of the current field
			if (c == '*')
			{
				width = __arg(args, int);
				if (width < 0)
				{
					flag_leftjustify = true;
					width = -width;
				}
				c = *++fmt;
			}
			else
			{
				while (c >= '0' && c <= '9')
				{
					width = width*10 + c - '0';
					c = *++fmt;
				}
			}
			// Get the precision
			int precision; // Precision of the current field
			if (c == '.')
			{
				precision = 0;
				c = *++fmt;
				if (c == '*')
				{
					precision = __arg(args, int);
					if (precision < 0) precision = -precision;
					c = *++fmt;
				}
				else
				{
					while (c >= '0' && c <= '9')
					{
						precision = precision*10 + c - '0';
						c = *++fmt;
					}
				}
			}
			else
				precision = -1;
			// Get the conversion type modifier
			bool flag_long; // True if "l" flag is present
			bool flag_longlong; // True if the "ll" flag is present
			if (c == 'l')
			{
				flag_long = true;
				c = *++fmt;
				if (c == 'l')
				{
					flag_longlong = true;
					c = *++fmt;
				}
				else
					flag_longlong = false;
			}
			else
				flag_long = flag_longlong = false;
			// Fetch the info entry for the field
			Info *info = &_info[0]; // Pointer to the appropriate info structure
			TYPE type = TYPE_INVALID; // Conversion paradigm
			int i;
			for (i = 0; i < __arrayStaticLength(_info); i++)
			{
				if (c == _info[i].Fmttype)
				{
					info = &_info[i];
					if (useExtended || (info->Flags & FLAG_INTERN) == 0) type = info->Type;
					else return;
					break;
				}
			}

			char prefix; // Prefix character.  "+" or "-" or " " or '\0'.
			uint64 longvalue; // Value for integer types
			double64 realvalue; // Value for real types
#ifndef OMIT_FLOATING_POINT
			int exp, e2; // exponent of real numbers
			int nsd; // Number of significant digits returned
			double rounder; // Used for rounding floating point values
			bool flag_dp; // True if decimal point should be shown
			bool flag_rtz; // True if trailing zeros should be removed
#endif

			// At this point, variables are initialized as follows:
			//   flag_alternateform          TRUE if a '#' is present.
			//   flag_altform2               TRUE if a '!' is present.
			//   flag_plussign               TRUE if a '+' is present.
			//   flag_leftjustify            TRUE if a '-' is present or if the field width was negative.
			//   flag_zeropad                TRUE if the width began with 0.
			//   flag_long                   TRUE if the letter 'l' (ell) prefixed the conversion character.
			//   flag_longlong               TRUE if the letter 'll' (ell ell) prefixed the conversion character.
			//   flag_blanksign              TRUE if a ' ' is present.
			//   width                       The specified field width.  This is always non-negative.  Zero is the default.
			//   precision                   The specified precision.  The default is -1.
			//   type                        The class of the conversion.
			//   info                        Pointer to the appropriate info struct.
			char *extra = nullptr; // Malloced memory used by some conversion
			char *out_; // Rendering buffer
			int outLength; // Size of the rendering buffer
			switch (type)
			{
			case TYPE_POINTER:
				flag_longlong = (sizeof(char *) == sizeof(int64));
				flag_long = (sizeof(char *) == sizeof(long int));
				// Fall through into the next case
			case TYPE_ORDINAL:
			case TYPE_RADIX:
				if (info->Flags & FLAG_SIGNED)
				{
					int64 v;
					if (flag_longlong) v = __arg(args, int64);
					else if (flag_long) v = __arg(args, long int);
					else v = __arg(args, int);
					if (v < 0)
					{
						longvalue = (v == SMALLEST_INT64 ? ((uint64)1)<<63 : -v);
						prefix = '-';
					}
					else
					{
						longvalue = v;
						if (flag_plussign) prefix = '+';
						else if (flag_blanksign) prefix = ' ';
						else prefix = '\0';
					}
				}
				else
				{
					if (flag_longlong) longvalue = __arg(args, uint64);
					else if (flag_long) longvalue = __arg(args, unsigned long int);
					else longvalue = __arg(args, unsigned int);
					prefix = 0;
				}
				if (longvalue == 0) flag_alternateform = false;
				if (flag_zeropad && precision < width - (prefix != '\0'))
					precision = width-(prefix!=0);
				if (precision < BUFSIZE-10)
				{
					outLength = BUFSIZE;
					out_ = buf;
				}
				else
				{
					outLength = precision + 10;
					out_ = extra = sqlite3Malloc(outLength);
					if (!out_)
					{
						b->MallocFailed = true;
						return;
					}
				}
				bufpt = &out_[outLength-1];
				if (type == TYPE_ORDINAL)
				{
					int x = (int)(longvalue % 10);
					if (x >= 4 || (longvalue/10)%10 == 1) x = 0;
					*(--bufpt) = _ord[x*2+1];
					*(--bufpt) = _ord[x*2];
				}
				{
					register const char *cset = &_digits[info->Charset]; // Use registers for speed
					register int base = info->Base;
					do // Convert to ascii
					{                                           
						*(--bufpt) = cset[longvalue % base];
						longvalue = longvalue / base;
					} while(longvalue > 0);
				}
				length = (int)(&out_[outLength-1]-bufpt);
				for (i = precision - length; i > 0; i--) *(--bufpt) = '0'; // Zero pad
				if (prefix) *(--bufpt) = prefix; // Add sign
				if (flag_alternateform && info->Prefix) // Add "0" or "0x"
				{
					char x;
					const char *pre = &_prefix[info->Prefix];
					for (; (x = (*pre)) != 0; pre++) *(--bufpt) = x;
				}
				length = (int)(&out_[outLength-1]-bufpt);
				break;
			case TYPE_FLOAT:
			case TYPE_EXP:
			case TYPE_GENERIC:
				realvalue = __arg(args, double);
#ifdef OMIT_FLOATING_POINT
				length = 0;
#else
				if (precision < 0) precision = 6; // Set default precision
				if (realvalue < 0.0)
				{
					realvalue = -realvalue;
					prefix = '-';
				}
				else
				{
					if (flag_plussign) prefix = '+';
					else if (flag_blanksign) prefix = ' ';
					else prefix = 0;
				}
				if (type == TYPE_GENERIC && precision > 0) precision--;
#if 0
				// Rounding works like BSD when the constant 0.4999 is used.  Wierd!
				for (i = precision, rounder = 0.4999; i > 0; i--, rounder *= 0.1);
#else
				// It makes more sense to use 0.5
				for (i = precision, rounder = 0.5; i > 0; i--, rounder *= 0.1) { }
#endif
				if (type == TYPE_FLOAT) realvalue += rounder;
				// Normalize realvalue to within 10.0 > realvalue >= 1.0
				exp = 0;
				if (_isNaN((double)realvalue))
				{
					bufpt = "NaN";
					length = 3;
					break;
				}
				if (realvalue > 0.0)
				{
					double64 scale = 1.0;
					while (realvalue >= 1e100*scale && exp <= 350) { scale *= 1e100;exp += 100; }
					while (realvalue >= 1e64*scale && exp <= 350) { scale *= 1e64; exp += 64; }
					while (realvalue >= 1e8*scale && exp <= 350) { scale *= 1e8; exp += 8; }
					while (realvalue >= 10.0*scale && exp <= 350) { scale *= 10.0; exp++; }
					realvalue /= scale;
					while (realvalue < 1e-8) { realvalue *= 1e8; exp -= 8; }
					while (realvalue < 1.0) { realvalue *= 10.0; exp--; }
					if (exp > 350)
					{
						if (prefix == '-') bufpt = "-Inf";
						else if (prefix == '+') bufpt = "+Inf";
						else bufpt = "Inf";
						length = _strlen30(bufpt);
						break;
					}
				}
				bufpt = buf;
				// If the field type is etGENERIC, then convert to either etEXP or etFLOAT, as appropriate.
				if (type != TYPE_FLOAT)
				{
					realvalue += rounder;
					if (realvalue >= 10.0) { realvalue *= 0.1; exp++; }
				}
				if (type == TYPE_GENERIC)
				{
					flag_rtz = !flag_alternateform;
					if (exp < -4 || exp > precision) type = TYPE_EXP;
					else { precision = precision - exp; type = TYPE_FLOAT; }
				}
				else
					flag_rtz = flag_altform2;
				e2 = (type == TYPE_EXP ? 0 : exp);
				if (e2+precision+width > BUFSIZE - 15)
				{
					bufpt = extra = sqlite3Malloc(e2+precision+width+15);
					if (!bufpt)
					{
						b->MallocFailed = true;
						return;
					}
				}
				out_ = bufpt;
				nsd = 16 + flag_altform2*10;
				flag_dp = (precision > 0) | flag_alternateform | flag_altform2;
				// The sign in front of the number
				if (prefix) *(bufpt++) = prefix;
				// Digits prior to the decimal point
				if (e2 < 0) *(bufpt++) = '0';
				else for (; e2 >= 0; e2--) *(bufpt++) = GetDigit(&realvalue, &nsd);
				// The decimal point
				if (flag_dp) *(bufpt++) = '.';
				// "0" digits after the decimal point but before the first significant digit of the number
				for (e2++; e2 < 0; precision--, e2++) { _assert(precision > 0); *(bufpt++) = '0'; }
				// Significant digits after the decimal point
				while ((precision--) > 0) *(bufpt++) = GetDigit(&realvalue, &nsd);
				// Remove trailing zeros and the "." if no digits follow the "."
				if (flag_rtz && flag_dp)
				{
					while (bufpt[-1] == '0') *(--bufpt) = 0;
					_assert(bufpt > out_);
					if (bufpt[-1] == '.')
					{
						if (flag_altform2) *(bufpt++) = '0';
						else *(--bufpt) = 0;
					}
				}
				// Add the "eNNN" suffix
				if (type == TYPE_EXP)
				{
					*(bufpt++) = _digits[info->Charset];
					if (exp < 0) { *(bufpt++) = '-'; exp = -exp; }
					else *(bufpt++) = '+';
					if (exp >= 100) { *(bufpt++) = (char)((exp/100)+'0'); exp %= 100; } // 100's digit
					*(bufpt++) = (char)(exp/10+'0'); // 10's digit
					*(bufpt++) = (char)(exp%10+'0'); // 1's digit
				}
				*bufpt = 0;

				// The converted number is in buf[] and zero terminated. Output it. Note that the number is in the usual order, not reversed as with integer conversions.
				length = (int)(bufpt-out_);
				bufpt = out_;

				// Special case:  Add leading zeros if the flag_zeropad flag is set and we are not left justified
				if (flag_zeropad && !flag_leftjustify && length < width)
				{
					int pad = width - length;
					for (i = width; i >= pad; i--) bufpt[i] = bufpt[i-pad];
					i = (prefix != '\0');
					while (pad--) bufpt[i++] = '0';
					length = width;
				}
#endif
				break;
			case TYPE_SIZE:
				*(__arg(args,int*)) = b->Size;
				length = width = 0;
				break;
			case TYPE_PERCENT:
				buf[0] = '%';
				bufpt = buf;
				length = 1;
				break;
			case TYPE_CHARX:
				c = __arg(args, int);
				buf[0] = (char)c;
				if (precision >= 0)
				{
					for (i = 1; i < precision; i++) buf[i] = (char)c;
					length = precision;
				}
				else length =1;
				bufpt = buf;
				break;
			case TYPE_STRING:
			case TYPE_DYNSTRING:
				bufpt = __arg(args, char*);
				if (bufpt == 0) bufpt = "";
				else if (type == TYPE_DYNSTRING) extra = bufpt;
				if (precision >= 0) for (length = 0; length < precision && bufpt[length]; length++) { }
				else length = _strlen30(bufpt);
				break;
			case TYPE_SQLESCAPE:
			case TYPE_SQLESCAPE2:
			case TYPE_SQLESCAPE3: {
				char q = (type == TYPE_SQLESCAPE3 ? '"' : '\''); // Quote character
				char *escarg = __arg(args, char*);
				bool isnull = (escarg == 0);
				if (isnull) escarg = (type == TYPE_SQLESCAPE2 ? "NULL" : "(NULL)");
				int k = precision;
				int j, n;
				char ch;
				for (i = n = 0; k != 0 && (ch = escarg[i]) != 0; i++, k--)
					if (ch == q) n++;
				bool needQuote = (!isnull && type == TYPE_SQLESCAPE2);
				n += i + 1 + needQuote*2;
				if (n > BUFSIZE)
				{
					bufpt = extra = sqlite3Malloc(n);
					if (!bufpt)
					{
						b->MallocFailed = true;
						return;
					}
				}
				else
					bufpt = buf;
				j = 0;
				if (needQuote) bufpt[j++] = q;
				k = i;
				for (i = 0; i < k; i++)
				{
					bufpt[j++] = ch = escarg[i];
					if (ch == q) bufpt[j++] = ch;
				}
				if (needQuote) bufpt[j++] = q;
				bufpt[j] = 0;
				length = j;
				// The precision in %q and %Q means how many input characters to consume, not the length of the output...
				// if (precision>=0 && precision<length) length = precision;
				break; }

			case TYPE_TOKEN: {
				Token *token = __arg(args, Token *);
				if (token) b->Append((const char *)token->z, token->n);
				length = width = 0;
				break; }
			case TYPE_SRCLIST: {
				SrcList *src = __arg(args, SrcList *);
				int k = __arg(args, int);
				SrcList::SrcListItem *item = &src->Ids[k];
				_assert(k >= 0 && k < src->Srcs);
				if (item->DatabaseName)
				{
					b->Append(item->DatabaseName, -1);
					b->Append(".", 1);
				}
				b->Append(item->Name, -1);
				length = width = 0;
				break; }
			default: {
				_assert(type == TYPE_INVALID);
				return; }
			}
			// The text of the conversion is pointed to by "bufpt" and is "length" characters long.  The field width is "width".  Do the output.
			if (!flag_leftjustify)
			{
				register int nspace = width-length;
				if (nspace > 0) AppendSpace(b, nspace);
			}
			if (length > 0) b->Append(bufpt, length);
			if (flag_leftjustify)
			{
				register int nspace = width-length;
				if (nspace > 0) AppendSpace(b, nspace);
			}
			if (extra != nullptr) SysEx::Free(extra);
		}
	}


#pragma endregion

#pragma region StringBuilder

	__device__ void StringBuilder::Append(const char *z, int length) //+ sqlite3StrAccumAppend
	{
		_assert(z != nullptr || length == 0);
		if (Overflowed | MallocFailed)
		{
			ASSERTCOVERAGE(Overflowed);
			ASSERTCOVERAGE(MallocFailed);
			return;
		}
		_assert(Text != nullptr || Index == 0);
		if (length < 0)
			length = _strlen30(z);
		if (length == 0 || SysEx_NEVER(z == nullptr))
			return;
		if (Index + length >= Size)
		{
			char *newText;
			if (!UseMalloc)
			{
				Overflowed = true;
				length = Size - Index - 1;
				if (length <= 0)
					return;
			}
			else
			{
				char *oldText = (Text == Base ? nullptr : Text);
				int64 newSize = Index;
				newSize += length + 1;
				if (newSize > MaxSize)
				{
					Reset();
					Overflowed = true;
					return;
				}
				else
					Size = (int)newSize;
				if (UseMalloc)
					newText = (char *)SysEx::TagRealloc(Ctx, oldText, Size);
				else
					newText = (char *)SysEx::Realloc(oldText, Size);
				if (newText)
				{
					if (oldText == nullptr && Index > 0) _memcpy(newText, Text, Index);
					Text = newText;
				}
				else
				{
					MallocFailed = true;
					Reset();
					return;
				}
			}
		}
		_assert(Text != nullptr);
		_memcpy(&Text[Index], z, length);
		Index += length;
	}

	__device__ char *StringBuilder::ToString() //+ sqlite3StrAccumFinish
	{
		if (Text)
		{
			Text[Index] = 0;
			if (UseMalloc && Text == Base)
			{
				if (UseMalloc)
					Text = (char *)SysEx::TagAlloc(Ctx, Index + 1);
				else
					Text = (char *)SysEx::Alloc(Index + 1);
				if (Text)
					_memcpy(Text, Base, Index + 1);
				else
					MallocFailed = true;
			}
		}
		return Text;
	}

	__device__ void StringBuilder::Reset() //+ sqlite3StrAccumReset
	{
		if (Text != Base)
		{
			if (UseMalloc == 1)
				SysEx::TagFree(Ctx, Text);
			else
				SysEx::Free(Text);
		}
		Text = nullptr;
	}

	__device__ void StringBuilder::Init(StringBuilder *b, char *text, int capacity, int maxSize) //+ sqlite3StrAccumInit
	{
		b->Text = b->Base = text;
		b->Ctx = nullptr;
		b->Index = 0;
		b->Size = capacity;
		b->MaxSize = maxSize;
		b->UseMalloc = 1;
		b->Overflowed = false;
		b->MallocFailed = false;
	}
#pragma endregion

#pragma region Printf

	/*
	** Print into memory obtained from sqliteMalloc().  Use the internal
	** %-conversion extensions.
	*/
	char *sqlite3VMPrintf(sqlite3 *db, const char *zFormat, va_list ap){
		char *z;
		char zBase[SQLITE_PRINT_BUF_SIZE];
		StrAccum acc;
		assert( db!=0 );
		sqlite3StrAccumInit(&acc, zBase, sizeof(zBase),
			db->aLimit[SQLITE_LIMIT_LENGTH]);
		acc.db = db;
		sqlite3VXPrintf(&acc, 1, zFormat, ap);
		z = sqlite3StrAccumFinish(&acc);
		if( acc.mallocFailed ){
			db->mallocFailed = 1;
		}
		return z;
	}

	/*
	** Print into memory obtained from sqliteMalloc().  Use the internal
	** %-conversion extensions.
	*/
	char *sqlite3MPrintf(sqlite3 *db, const char *zFormat, ...){
		va_list ap;
		char *z;
		va_start(ap, zFormat);
		z = sqlite3VMPrintf(db, zFormat, ap);
		va_end(ap);
		return z;
	}

	/*
	** Like sqlite3MPrintf(), but call sqlite3DbFree() on zStr after formatting
	** the string and before returnning.  This routine is intended to be used
	** to modify an existing string.  For example:
	**
	**       x = sqlite3MPrintf(db, x, "prefix %s suffix", x);
	**
	*/
	char *sqlite3MAppendf(sqlite3 *db, char *zStr, const char *zFormat, ...){
		va_list ap;
		char *z;
		va_start(ap, zFormat);
		z = sqlite3VMPrintf(db, zFormat, ap);
		va_end(ap);
		sqlite3DbFree(db, zStr);
		return z;
	}

	/*
	** Print into memory obtained from sqlite3_malloc().  Omit the internal
	** %-conversion extensions.
	*/
	char *sqlite3_vmprintf(const char *zFormat, va_list ap){
		char *z;
		char zBase[SQLITE_PRINT_BUF_SIZE];
		StrAccum acc;
#ifndef SQLITE_OMIT_AUTOINIT
		if( sqlite3_initialize() ) return 0;
#endif
		sqlite3StrAccumInit(&acc, zBase, sizeof(zBase), SQLITE_MAX_LENGTH);
		acc.useMalloc = 2;
		sqlite3VXPrintf(&acc, 0, zFormat, ap);
		z = sqlite3StrAccumFinish(&acc);
		return z;
	}

	/*
	** Print into memory obtained from sqlite3_malloc()().  Omit the internal
	** %-conversion extensions.
	*/
	char *sqlite3_mprintf(const char *zFormat, ...){
		va_list ap;
		char *z;
#ifndef SQLITE_OMIT_AUTOINIT
		if( sqlite3_initialize() ) return 0;
#endif
		va_start(ap, zFormat);
		z = sqlite3_vmprintf(zFormat, ap);
		va_end(ap);
		return z;
	}

	/*
	** sqlite3_snprintf() works like snprintf() except that it ignores the
	** current locale settings.  This is important for SQLite because we
	** are not able to use a "," as the decimal point in place of "." as
	** specified by some locales.
	**
	** Oops:  The first two arguments of sqlite3_snprintf() are backwards
	** from the snprintf() standard.  Unfortunately, it is too late to change
	** this without breaking compatibility, so we just have to live with the
	** mistake.
	**
	** sqlite3_vsnprintf() is the varargs version.
	*/
	char *sqlite3_vsnprintf(int n, char *zBuf, const char *zFormat, va_list ap){
		StrAccum acc;
		if( n<=0 ) return zBuf;
		sqlite3StrAccumInit(&acc, zBuf, n, 0);
		acc.useMalloc = 0;
		sqlite3VXPrintf(&acc, 0, zFormat, ap);
		return sqlite3StrAccumFinish(&acc);
	}
	char *sqlite3_snprintf(int n, char *zBuf, const char *zFormat, ...){
		char *z;
		va_list ap;
		va_start(ap,zFormat);
		z = sqlite3_vsnprintf(n, zBuf, zFormat, ap);
		va_end(ap);
		return z;
	}

	/*
	** This is the routine that actually formats the sqlite3_log() message.
	** We house it in a separate routine from sqlite3_log() to avoid using
	** stack space on small-stack systems when logging is disabled.
	**
	** sqlite3_log() must render into a static buffer.  It cannot dynamically
	** allocate memory because it might be called while the memory allocator
	** mutex is held.
	*/
	static void renderLogMsg(int iErrCode, const char *zFormat, va_list ap){
		StrAccum acc;                          /* String accumulator */
		char zMsg[SQLITE_PRINT_BUF_SIZE*3];    /* Complete log message */

		sqlite3StrAccumInit(&acc, zMsg, sizeof(zMsg), 0);
		acc.useMalloc = 0;
		sqlite3VXPrintf(&acc, 0, zFormat, ap);
		sqlite3GlobalConfig.xLog(sqlite3GlobalConfig.pLogArg, iErrCode,
			sqlite3StrAccumFinish(&acc));
	}

	/*
	** Format and write a message to the log if logging is enabled.
	*/
	void sqlite3_log(int iErrCode, const char *zFormat, ...){
		va_list ap;                             /* Vararg list */
		if( sqlite3GlobalConfig.xLog ){
			va_start(ap, zFormat);
			renderLogMsg(iErrCode, zFormat, ap);
			va_end(ap);
		}
	}

#ifdef _DEBUG
	void sqlite3DebugPrintf(const char *zFormat, ...){
		va_list ap;
		StrAccum acc;
		char zBuf[500];
		sqlite3StrAccumInit(&acc, zBuf, sizeof(zBuf), 0);
		acc.useMalloc = 0;
		va_start(ap,zFormat);
		sqlite3VXPrintf(&acc, 0, zFormat, ap);
		va_end(ap);
		sqlite3StrAccumFinish(&acc);
		fprintf(stdout,"%s", zBuf);
		fflush(stdout);
	}
#endif

#ifndef OMIT_TRACE
	void sqlite3XPrintf(StrAccum *p, const char *zFormat, ...){
		va_list ap;
		va_start(ap,zFormat);
		sqlite3VXPrintf(p, 1, zFormat, ap);
		va_end(ap);
	}
#endif
#pragma endregion

}}


