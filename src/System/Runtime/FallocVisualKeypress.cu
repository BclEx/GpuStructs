#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "Falloc.cu.h"

__device__ void *_testObj;
__device__ fallocCtx *_testCtx;

__global__ static void Keypress(fallocHeap *heap, unsigned char key)
{
	switch (key)
	{
	case 'a':
		_testObj = fallocGetBlock(heap);
		break;
	case 'b':
		fallocFreeBlock(heap, _testObj);
		break;
	case 'x':
		_testCtx = fallocCreateCtx(heap);
		break;
	case 'y':
		char *testString = (char *)falloc(_testCtx, 10);
		int *testInteger = falloc<int>(_testCtx);
		break;
	case 'z':
		fallocDisposeCtx(_testCtx);
		break;
	}
}

__host__ void LaunchFallocKeypress(fallocHeap *heap, unsigned char key)
{
	dim3 heapBlock(1, 1, 1);
	dim3 heapGrid(1, 1, 1);
	Keypress<<<heapGrid, heapBlock>>>(heap, key);
}
