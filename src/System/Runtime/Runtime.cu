#define nullptr NULL
//#define __THROW *(int*)0=0;
#if __CUDA_ARCH__ == 100 
#error Atomics only used with > sm_10 architecture
#endif

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
//#include <malloc.h>
#include <string.h>

#define RUNTIME_UNRESTRICTED -1

typedef struct __align__(8)
{
	int threadid; // RUNTIME_UNRESTRICTED for unrestricted
	int blockid;  // RUNTIME_UNRESTRICTED for unrestricted
} runtimeRestriction;

typedef struct __align__(8)
{
	volatile char *blockPtr; // current atomically-incremented non-wrapped offset
	void *reserved;
	runtimeRestriction restriction;
	size_t blockSize;
	size_t blocksLength; // size of circular buffer (set up by host)
	char *blocks; // start of circular buffer (set up by host)
} runtimeHeap;

typedef struct __align__(8)
{
	unsigned short magic;		// magic number says we're valid
	unsigned short type;		// type of block
	unsigned short fmtoffset;	// offset of fmt string into buffer
	unsigned short blockid;		// block ID of author
	unsigned short threadid;	// thread ID of author
} runtimeBlockHeader;

__device__ static runtimeHeap *__runtimeHeap = nullptr;
__device__ static void setRuntimeHeap(runtimeHeap *heap) { __runtimeHeap = heap; }

///////////////////////////////////////////////////////////////////////////////
// HEAP
#pragma region HEAP

#define RUNTIME_MAGIC (unsigned short)0xC811
#define RUNTIME_ALIGNSIZE sizeof(long long)
#define RUNTIMETYPE_PRINTF 1
#define RUNTIMETYPE_ASSERT 2
#define RUNTIMETYPE_THROW 2

__device__ static char *moveNextPtr()
{
	if (!__runtimeHeap) __THROW;
	// thread/block restriction check
	runtimeRestriction restriction = __runtimeHeap->restriction;
	if (restriction.blockid != RUNTIME_UNRESTRICTED && restriction.blockid != (blockIdx.x + gridDim.x*blockIdx.y))
		return nullptr;
	if (restriction.threadid != RUNTIME_UNRESTRICTED && restriction.threadid != (threadIdx.x + blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z))
		return nullptr;
	// advance pointer
	char *blocks = __runtimeHeap->blocks;
	size_t offset = atomicAdd((unsigned int *)&__runtimeHeap->blockPtr, __runtimeHeap->blockSize) - (size_t)blocks;
	offset %= __runtimeHeap->blocksLength;
	return blocks + offset;
}

__device__ static void runtimeRestrict(int threadid, int blockid)
{
	int threadMax = blockDim.x * blockDim.y * blockDim.z;
	if ((threadid < threadMax && threadid >= 0) || threadid == RUNTIME_UNRESTRICTED)
		__runtimeHeap->restriction.threadid = threadid;
	int blockMax = gridDim.x * gridDim.y;
	if ((blockid < blockMax && blockid >= 0) || blockid == RUNTIME_UNRESTRICTED)
		__runtimeHeap->restriction.blockid = blockid;
}

__inline__ __device__ static void writeBlockHeader(unsigned short type, char *ptr, char *fmtptr)
{
	runtimeBlockHeader header;
	header.magic = RUNTIME_MAGIC;
	header.type = type;
	header.fmtoffset = (unsigned short)(fmtptr - ptr);
	header.blockid = gridDim.x*blockIdx.y + blockIdx.x;
	header.threadid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
	*(runtimeBlockHeader *)(void *)ptr = header;
}

__device__ static char *writeString(char *dest, const char *src, int n, char *end)
{
	// initialization and overflow check
	if (!dest || src != 0 || dest >= end)
		return nullptr;
	// prepare to write the length specifier. We're guaranteed to have at least "RUNTIME_ALIGNSIZE" bytes left because we only write out in
	// chunks that size, and blockSize is aligned with RUNTIME_ALIGNSIZE.
	int *lenptr = (int *)(void *)dest;
	int len = 0;
	dest += RUNTIME_ALIGNSIZE;
	// now copy the string
	while (n--)
	{
		if (dest >= end) // overflow check
			break;
		len++;
		*dest++ = *src;
		if (*src++ == '\0')
			break;
	}
	// now write out the padding bytes, and we have our length.
	while (dest < end && ((long)dest & (RUNTIME_ALIGNSIZE - 1)) != 0)
	{
		len++;
		*dest++ = 0;
	}
	*lenptr = len;
	return (dest < end ? dest : nullptr); // overflow means return nullptr
}

__device__ static char *copyArg(char *ptr, const char *arg, char *end)
{
	// initialization check
	if (!ptr || !arg)
		return nullptr;
	// strncpy does all our work. We just terminate.
	if ((ptr = writeString(ptr, arg, __runtimeHeap->blockSize, end)) != nullptr)
		*ptr = 0;
	return ptr;
}

template <typename T>
__device__ static char *copyArg(char *ptr, T &arg, char *end)
{
	// initialization and overflow check. Alignment rules mean that we're at least CUPRINTF_ALIGN_SIZE away from "end", so we only need to check that one offset.
	if (!ptr || (ptr + RUNTIME_ALIGNSIZE) >= end)
		return nullptr;
	// write the length and argument
	*(int *)(void *)ptr = sizeof(arg);
	ptr += RUNTIME_ALIGNSIZE;
	*(T *)(void *)ptr = arg;
	ptr += RUNTIME_ALIGNSIZE;
	*ptr = 0;
	return ptr;
}

#pragma endregion


//////////////////////
// PRINTF
#pragma region PRINTF

#define PRINTF_PREAMBLE \
	char *start, *end, *bufptr, *fmtstart; \
	if ((start = moveNextPtr()) == nullptr) return 0; \
	end = start + __runtimeHeap->blockSize; \
	bufptr = start + sizeof(runtimeBlockHeader);
#define PRINTF_ARG(argname) \
	bufptr = copyArg(bufptr, argname, end);
#define PRINTF_POSTAMBLE \
	fmtstart = bufptr; \
	end = writeString(bufptr, fmt, __runtimeHeap->blockSize, end); \
	writeBlockHeader(RUNTIMETYPE_PRINTF, start, (end ? fmtstart : nullptr)); \
	return (end ? (int)(end - start) : 0);

__device__ static int __printf(const char *fmt)
{
	PRINTF_PREAMBLE;
	PRINTF_POSTAMBLE;
}
template <typename T1> __device__ static int __printf(const char *fmt, T1 arg1)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2, T3 arg3)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_ARG(arg3);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_ARG(arg3);
	PRINTF_ARG(arg4);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_ARG(arg3);
	PRINTF_ARG(arg4);
	PRINTF_ARG(arg5);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_ARG(arg3);
	PRINTF_ARG(arg4);
	PRINTF_ARG(arg5);
	PRINTF_ARG(arg6);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6, typename T7> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6, T7 arg7)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_ARG(arg3);
	PRINTF_ARG(arg4);
	PRINTF_ARG(arg5);
	PRINTF_ARG(arg6);
	PRINTF_ARG(arg7);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6, typename T7, typename T8> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6, T7 arg7, T8 arg8)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_ARG(arg3);
	PRINTF_ARG(arg4);
	PRINTF_ARG(arg5);
	PRINTF_ARG(arg6);
	PRINTF_ARG(arg7);
	PRINTF_ARG(arg8);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6, typename T7, typename T8, typename T9> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6, T7 arg7, T8 arg8, T9 arg9)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_ARG(arg3);
	PRINTF_ARG(arg4);
	PRINTF_ARG(arg5);
	PRINTF_ARG(arg6);
	PRINTF_ARG(arg7);
	PRINTF_ARG(arg8);
	PRINTF_ARG(arg9);
	PRINTF_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6, typename T7, typename T8, typename T9, typename T10> __device__ static int __printf(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4, T5 arg5, T6 arg6, T7 arg7, T8 arg8, T9 arg9, T10 arg10)
{
	PRINTF_PREAMBLE;
	PRINTF_ARG(arg1);
	PRINTF_ARG(arg2);
	PRINTF_ARG(arg3);
	PRINTF_ARG(arg4);
	PRINTF_ARG(arg5);
	PRINTF_ARG(arg6);
	PRINTF_ARG(arg7);
	PRINTF_ARG(arg8);
	PRINTF_ARG(arg9);
	PRINTF_ARG(arg10);
	PRINTF_POSTAMBLE;
}

#undef PRINTF_PREAMBLE
#undef PRINTF_ARG
#undef PRINTF_POSTAMBLE

#pragma endregion


//////////////////////
// ASSERT
#pragma region ASSERT

#define ASSERT_PREAMBLE \
	char *start, *end, *bufptr, *fmtstart; \
	if ((start = moveNextPtr()) == nullptr) return; \
	end = start + __runtimeHeap->blockSize; \
	bufptr = start + sizeof(runtimeBlockHeader);
#define ASSERT_ARG(argname) \
	bufptr = copyArg(bufptr, argname, end);
#define ASSERT_POSTAMBLE \
	fmtstart = bufptr; \
	end = writeString(bufptr, fmt, __runtimeHeap->blockSize, end); \
	writeBlockHeader(RUNTIMETYPE_ASSERT, start, (end ? fmtstart : nullptr));

__device__ static void __assertD(const bool condition)
{
	const char *fmt = nullptr;
	if (condition)
	{
		ASSERT_PREAMBLE;
		ASSERT_POSTAMBLE;
	}
}
__device__ static void __assertD(const bool condition, const char *fmt)
{
	if (condition)
	{
		ASSERT_PREAMBLE;
		ASSERT_POSTAMBLE;
	}
}

#undef ASSERT_PREAMBLE
#undef ASSERT_ARG
#undef ASSERT_POSTAMBLE

#pragma endregion

//////////////////////
// THROW
#pragma region THROW

#define THROW_PREAMBLE \
	char *start, *end, *bufptr, *fmtstart; \
	if ((start = moveNextPtr()) == nullptr) return; \
	end = start + __runtimeHeap->blockSize; \
	bufptr = start + sizeof(runtimeBlockHeader);
#define THROW_ARG(argname) \
	bufptr = copyArg(bufptr, argname, end);
#define THROW_POSTAMBLE \
	fmtstart = bufptr; \
	end = writeString(bufptr, fmt, __runtimeHeap->blockSize, end); \
	writeBlockHeader(RUNTIMETYPE_THROW, start, (end ? fmtstart : nullptr)); \
	__THROW;

__device__ static void __throw(const char *fmt)
{
	THROW_PREAMBLE;
	THROW_POSTAMBLE;
}
template <typename T1> __device__ static void __throw(const char *fmt, T1 arg1)
{
	THROW_PREAMBLE;
	THROW_ARG(arg1);
	THROW_POSTAMBLE;
}
template <typename T1, typename T2> __device__ static void __throw(const char *fmt, T1 arg1, T2 arg2)
{
	THROW_PREAMBLE;
	THROW_ARG(arg1);
	THROW_ARG(arg2);
	THROW_POSTAMBLE;
}
template <typename T1, typename T2, typename T3> __device__ static void __throw(const char *fmt, T1 arg1, T2 arg2, T3 arg3)
{
	THROW_PREAMBLE;
	THROW_ARG(arg1);
	THROW_ARG(arg2);
	THROW_ARG(arg3);
	THROW_POSTAMBLE;
}
template <typename T1, typename T2, typename T3, typename T4> __device__ static void __throw(const char *fmt, T1 arg1, T2 arg2, T3 arg3, T4 arg4)
{
	THROW_PREAMBLE;
	THROW_ARG(arg1);
	THROW_ARG(arg2);
	THROW_ARG(arg3);
	THROW_ARG(arg4);
	THROW_POSTAMBLE;
}

#undef THROW_PREAMBLE
#undef THROW_ARG
#undef THROW_POSTAMBLE

#pragma endregion
