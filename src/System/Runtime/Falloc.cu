#include "hip/hip_runtime.h"
//[GL VBO] http://3dgep.com/?p=2596
#if __CUDA_ARCH__ == 100
#error Atomics only used with > sm_10 architecture
#elif __CUDA_ARCH__ < 200
#define STATIC static
#else
#define STATIC
#endif
#include "hip/hip_runtime_api.h"
#include <malloc.h>
#include <string.h>

typedef struct __align__(8)
{
	unsigned short magic;		// magic number says we're valid
	unsigned short count;		// number of blocks in sequence
	unsigned short blockid;		// block ID of author
	unsigned short threadid;	// thread ID of author
} fallocBlockHeader;

typedef struct __align__(8)
{
	fallocBlockHeader *block;	// block reference
	unsigned short blockid;		// block ID of author
	unsigned short threadid;	// thread ID of author
} fallocBlockRef;

typedef struct __align__(8)
{
	void *reserved;
	size_t blockSize;
	size_t blocksLength;
	size_t blockRefsLength; // Size of circular buffer (set up by host)
	fallocBlockRef *blockRefs; // Start of circular buffer (set up by host)
	volatile fallocBlockRef *freeBlockPtr; // Current atomically-incremented non-wrapped offset
	volatile fallocBlockRef *retnBlockPtr; // Current atomically-incremented non-wrapped offset
	char *blocks;
} fallocHeap;

///////////////////////////////////////////////////////////////////////////////
// HEAP
#pragma region HEAP

#define FALLOC_MAGIC (unsigned short)0x3412 // All our headers are prefixed with a magic number so we know they're ours

__inline__ __device__ static void writeBlockRef(fallocBlockRef *ref, fallocBlockHeader *block)
{
	ref->block = block;
	ref->blockid = gridDim.x*blockIdx.y + blockIdx.x;
	ref->threadid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
}

__inline__ __device__ static void writeBlockHeader(fallocBlockHeader *hdr, unsigned short count)
{
	fallocBlockHeader header;
	header.magic = FALLOC_MAGIC;
	header.count = count;
	header.blockid = gridDim.x*blockIdx.y + blockIdx.x;
	header.threadid = blockDim.x*blockDim.y*threadIdx.z + blockDim.x*threadIdx.y + threadIdx.x;
	*hdr = header;
}

static __inline__ __device__ void *fallocGetBlock(fallocHeap *heap)
{
	// advance circular buffer
	fallocBlockRef *blockRefs = heap->blockRefs;
	size_t offset = atomicAdd((unsigned int *)&heap->freeBlockPtr, sizeof(fallocBlockRef)) - (size_t)blockRefs;
	offset %= heap->blockRefsLength;
	fallocBlockRef *blockRef = (fallocBlockRef *)((char *)blockRefs + offset);
	fallocBlockHeader *block = blockRef->block;
	writeBlockHeader(block, 1);
	blockRef->block = nullptr;
	return (void *)((char *)block + sizeof(fallocBlockHeader));
}

static __inline__ __device__ void fallocFreeBlock(fallocHeap *heap, void *obj)
{
	fallocBlockHeader *block = (fallocBlockHeader *)((char *)obj - sizeof(fallocBlockHeader));
	if (block->magic != FALLOC_MAGIC || block->count > 1) __THROW;// bad magic or not a singular block
	// advance circular buffer
	fallocBlockRef *blockRefs = heap->blockRefs;
	size_t offset = atomicAdd((unsigned int *)&heap->retnBlockPtr, sizeof(fallocBlockRef)) - (size_t)blockRefs;
	offset %= heap->blockRefsLength;
	writeBlockRef((fallocBlockRef *)((char *)blockRefs + offset), block);
	block->magic = 0;
}

/*
__device__ inline void *fallocGetBlocks(fallocHeap *heap, size_t length, size_t *allocLength = nullptr)
{
	if (threadIdx.x || threadIdx.y || threadIdx.z) __THROW;
	size_t blockSize = heap->blockSize;
	// fix up length to be a multiple of blockSize
	if (length % blockSize)
		length += blockSize - (length % blockSize);
	// set length, if requested
	if (allocLength)
		*allocLength = length - sizeof(fallocBlockHeader);
	size_t blocks = (size_t)(length / blockSize);
	if (blocks > heap->blocks) __THROW;
	// single, equals: fallocGetBlock
	if (blocks == 1)
		return fallocGetBlock(heap);
	// multiple, find a contiguous chuck
	size_t index = blocks;
	volatile fallocBlockHeader* block;
	volatile fallocBlockHeader* endBlock = (fallocBlockHeader*)((__int8*)heap + sizeof(fallocHeap) + (blockSize * heap->blocks));
	{ // critical
		for (block = (fallocBlockHeader*)((__int8*)heap + sizeof(fallocHeap)); index && block < endBlock; block = (fallocBlockHeader*)((__int8*)block + (blockSize * block->count)))
		{
			if (block->magic != FALLOC_MAGIC)
				__THROW;
			index = (block->next ? index - 1 : blocks);
		}
		if (index)
			return nullptr;
		// found chuck, remove from blockRefs
		endBlock = block;
		block = (fallocBlockHeader*)((__int8*)block - (blockSize * blocks));
		for (volatile fallocBlockHeader* chunk2 = heap->blockRefs; chunk2; chunk2 = chunk2->next)
			if (chunk2 >= block && chunk2 <= endBlock)
				chunk2->next = (chunk2->next ? chunk2->next->next : nullptr);
		block->count = blocks;
		block->next = nullptr;
	}
	return (void*)((__int8*)block + sizeof(fallocBlockHeader));
}


__device__ inline void fallocFreeBlocks(fallocHeap *heap, void *obj)
{
	volatile fallocBlockHeader* block = (fallocBlockHeader*)((__int8*)obj - sizeof(fallocBlockHeader));
	if (block->magic != FALLOC_MAGIC)
		__THROW;
	size_t blocks = block->count;
	// single, equals: fallocFreeChunk
	if (blocks == 1)
	{
		{ // critical
			block->next = heap->blockRefs;
			heap->blockRefs = block;
		}
		return;
	}
	// retag blocks
	size_t blockSize = heap->blockSize;
	block->count = 1;
	while (blocks-- > 1)
	{
		block = block->next = (fallocBlockHeader*)((__int8*)block + sizeof(fallocBlockHeader) + blockSize);
		block->magic = FALLOC_MAGIC;
		block->count = 1;
		block->reserved = nullptr;
	}
	{ // critical
		block->next = heap->blockRefs;
		heap->blockRefs = block;
	}
}
*/
#pragma endregion


//////////////////////
// CONTEXT
#pragma region CONTEXT

const static int FALLOCNODE_SLACK = 0x10;
#define FALLOCNODE_MAGIC (unsigned short)0x7856 // All our headers are prefixed with a magic number so we know they're ours

typedef struct _cuFallocNode
{
	struct _cuFallocNode *next;
	struct _cuFallocNode *nextAvailable;
	unsigned short freeOffset;
	unsigned short magic;
} fallocNode;

typedef struct _cuFallocContext
{
	fallocNode node;
	fallocNode *nodes;
	fallocNode *availableNodes;
	fallocHeap *heap;
	size_t HEAPBLOCK_SIZE;
} fallocCtx;

STATIC __device__ fallocCtx *fallocCreateCtx(fallocHeap *heap)
{
	size_t blockSize = heap->blockSize;
	if (sizeof(fallocCtx) > blockSize) __THROW;
	fallocCtx *ctx = (fallocCtx *)fallocGetBlock(heap);
	if (!ctx)
		return nullptr;
	ctx->heap = heap;
	unsigned short freeOffset = ctx->node.freeOffset = sizeof(fallocCtx);
	ctx->node.magic = FALLOCNODE_MAGIC;
	ctx->node.next = nullptr; ctx->nodes = (fallocNode *)ctx;
	ctx->node.nextAvailable = nullptr; ctx->availableNodes = (fallocNode *)ctx;
	// close node
	if ((freeOffset + FALLOCNODE_SLACK) > blockSize)
		ctx->availableNodes = nullptr;
	return ctx;
}

STATIC __device__ void fallocDisposeCtx(fallocCtx *ctx)
{
	fallocHeap *heap = ctx->heap;
	for (fallocNode *node = ctx->nodes; node; node = node->next)
		fallocFreeBlock(heap, node);
}

STATIC __device__ void *falloc(fallocCtx *ctx, unsigned short bytes, bool alloc = true)
{
	if (bytes > (ctx->HEAPBLOCK_SIZE - sizeof(fallocCtx))) __THROW;
	// find or add available node
	fallocNode *node;
	unsigned short freeOffset;
	unsigned char hasFreeSpace;
	fallocNode *lastNode;
	for (lastNode = (fallocNode *)ctx, node = ctx->availableNodes; node; lastNode = node, node = (alloc ? node->nextAvailable : node->next))
		if (hasFreeSpace = ((freeOffset = (node->freeOffset + bytes)) <= ctx->HEAPBLOCK_SIZE))
			break;
	if (!node || !hasFreeSpace) {
		// add node
		node = (fallocNode *)fallocGetBlock(ctx->heap);
		if (!node) __THROW;
		freeOffset = node->freeOffset = sizeof(fallocNode); 
		freeOffset += bytes;
		node->magic = FALLOCNODE_MAGIC;
		node->next = ctx->nodes; ctx->nodes = node;
		node->nextAvailable = (alloc ? ctx->availableNodes : nullptr); ctx->availableNodes = node;
	}
	//
	void *obj = (__int8 *)node + node->freeOffset;
	node->freeOffset = freeOffset;
	// close node
	if (alloc && ((freeOffset + FALLOCNODE_SLACK) > ctx->HEAPBLOCK_SIZE)) {
		if (lastNode == (fallocNode *)ctx)
			ctx->availableNodes = node->nextAvailable;
		else
			lastNode->nextAvailable = node->nextAvailable;
		node->nextAvailable = nullptr;
	}
	return obj;
}

STATIC __device__ void *fallocRetract(fallocCtx *ctx, unsigned short bytes)
{
	fallocNode *node = ctx->availableNodes;
	int freeOffset = (int)node->freeOffset - bytes;
	// multi node, retract node
	if (node != &ctx->node && freeOffset < sizeof(fallocNode)) {
		node->freeOffset = sizeof(fallocNode);
		// search for previous node
		fallocNode *lastNode;
		for (lastNode = (fallocNode *)ctx, node = ctx->nodes; node; lastNode = node, node = node->next)
			if (node == ctx->availableNodes)
				break;
		node = ctx->availableNodes = lastNode;
		freeOffset = (int)node->freeOffset - bytes;
	}
	// first node && !overflow
	if (node == &ctx->node && freeOffset < sizeof(fallocCtx)) __THROW;
	node->freeOffset = (unsigned short)freeOffset;
	return (__int8 *)node + freeOffset;
}

static __inline__ __device__ void fallocMark(fallocCtx *ctx, void *&mark, unsigned short &mark2)
{
	mark = ctx->availableNodes; mark2 = ctx->availableNodes->freeOffset;
}

static __inline__ __device__ bool fallocAtMark(fallocCtx *ctx, void *mark, unsigned short mark2)
{
	return (mark == ctx->availableNodes && mark2 == ctx->availableNodes->freeOffset);
}

template <typename T> __device__ T* falloc(fallocCtx *ctx) { return (T *)falloc(ctx, sizeof(T), true); }
template <typename T> __device__ void fallocPush(fallocCtx *ctx, T t) { *((T *)falloc(ctx, sizeof(T), false)) = t; }
template <typename T> __device__ T fallocPop(fallocCtx *ctx) { return *((T *)fallocRetract(ctx, sizeof(T))); }

#pragma endregion