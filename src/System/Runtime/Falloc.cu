#if __CUDA_ARCH__ == 100 
#error Atomics only used with > sm_10 architecture
#endif
#include "hip/hip_runtime_api.h"
#include <malloc.h>
#include <string.h>

typedef struct __align__(8) _cuFallocBlock
{
	unsigned short magic;
	unsigned short count;
	struct _cuFallocBlock* next;
	void* reserved;
} fallocBlock;

typedef struct __align__(8) _cuFallocHeap
{
	size_t blockSize;
	size_t blocks;
	size_t offset;
	size_t freeBlocksSize; // Size of circular buffer (set up by host)
	fallocBlock** freeBlocks; // Start of circular buffer (set up by host)
	volatile fallocBlock** freeBlocksPtr; // Current atomically-incremented non-wrapped offset
	void* reserved;
} fallocHeap;


///////////////////////////////////////////////////////////////////////////////
// DEVICE SIDE

const static int FALLOCNODE_SLACK = 0x10;
#define FALLOC_MAGIC (unsigned short)0x3412 // All our headers are prefixed with a magic number so we know they're ours
#define FALLOCNODE_MAGIC (unsigned short)0x7856 // All our headers are prefixed with a magic number so we know they're ours

typedef struct _cuFallocNode
{
	struct _cuFallocNode* next;
	struct _cuFallocNode* nextAvailable;
	unsigned short freeOffset;
	unsigned short magic;
} fallocNode;

typedef struct _cuFallocContext
{
	fallocNode node;
	fallocNode* nodes;
	fallocNode* availableNodes;
	fallocHeap* heap;
} fallocCtx;

__device__ void fallocInit(fallocHeap* heap)
{
	if (threadIdx.x || threadIdx.y || threadIdx.z) return;
	size_t blocks = heap->blocks;
	if (!blocks)
		__THROW;
	fallocBlock** freeBlocks = heap->freeBlocks;
	size_t blockSize = heap->blockSize;
	// preset all blocks
	fallocBlock* block = (fallocBlock*)((__int8*)heap + heap->offset);
	block->magic = FALLOC_MAGIC;
	block->count = 1;
	block->reserved = nullptr;
	while (blocks-- > 1)
	{
		block = *freeBlocks++ = block->next = (fallocBlock*)((__int8 *)block + blockSize);
		block->magic = FALLOC_MAGIC;
		block->count = 1;
		block->reserved = nullptr;
	}
	block->next = nullptr;
	heap->freeBlocksPtr = freeBlocks;
}

__device__ inline void* fallocGetBlock(fallocHeap* heap)
{
	if (threadIdx.x || threadIdx.y || threadIdx.z) __THROW;
	volatile fallocBlock* block = heap->freeBlocks;
	if (!block)
		return nullptr;
	{ // critical
		heap->freeBlocks = block->next;
		block->next = nullptr;
	}
	return (void*)((__int8*)block + sizeof(fallocBlock));
}

__device__ inline void* fallocGetBlocks(fallocHeap* heap, size_t length, size_t* allocLength = nullptr)
{
	if (threadIdx.x || threadIdx.y || threadIdx.z) __THROW;
	size_t blockSize = heap->blockSize;
	// fix up length to be a multiple of blockSize
	length = (length < blockSize ? blockSize : length);
	if (length % blockSize)
		length += blockSize - (length % blockSize);
	// set length, if requested
	if (allocLength)
		*allocLength = length - sizeof(fallocBlock);
	size_t blocks = (size_t)(length / blockSize);
	if (blocks > heap->blocks)
		__THROW;
	// single, equals: fallocGetBlock
	if (blocks == 1)
		return fallocGetBlock(heap);
	// multiple, find a contiguous chuck
	size_t index = blocks;
	volatile fallocBlock* block;
	volatile fallocBlock* endBlock = (fallocBlock*)((__int8*)heap + sizeof(fallocHeap) + (blockSize * heap->blocks));
	{ // critical
		for (block = (fallocBlock*)((__int8*)heap + sizeof(fallocHeap)); index && block < endBlock; block = (fallocBlock*)((__int8*)block + (blockSize * block->count)))
		{
			if (block->magic != FALLOC_MAGIC)
				__THROW;
			index = (block->next ? index - 1 : blocks);
		}
		if (index)
			return nullptr;
		// found chuck, remove from freeBlocks
		endBlock = block;
		block = (fallocBlock*)((__int8*)block - (blockSize * blocks));
		for (volatile fallocBlock* chunk2 = heap->freeBlocks; chunk2; chunk2 = chunk2->next)
			if (chunk2 >= block && chunk2 <= endBlock)
				chunk2->next = (chunk2->next ? chunk2->next->next : nullptr);
		block->count = blocks;
		block->next = nullptr;
	}
	return (void*)((__int8*)block + sizeof(fallocBlock));
}

__device__ inline void fallocFreeBlock(fallocHeap* heap, void* obj)
{
	if (threadIdx.x || threadIdx.y || threadIdx.z) __THROW;
	volatile fallocBlock* block = (fallocBlock *)((__int8 *)obj - (int)sizeof(fallocBlock));
	if (block->magic != FALLOC_MAGIC || block->count > 1)
		__THROW;
	{ // critical
		block->next = heap->freeBlocks;
		heap->freeBlocks = block;
	}
}

__device__ inline void fallocFreeBlocks(fallocHeap* heap, void* obj)
{
	volatile fallocBlock* block = (fallocBlock*)((__int8*)obj - sizeof(fallocBlock));
	if (block->magic != FALLOC_MAGIC)
		__THROW;
	size_t blocks = block->count;
	// single, equals: fallocFreeChunk
	if (blocks == 1)
	{
		{ // critical
			block->next = heap->freeBlocks;
			heap->freeBlocks = block;
		}
		return;
	}
	// retag blocks
	size_t blockSize = heap->blockSize;
	block->count = 1;
	while (blocks-- > 1)
	{
		block = block->next = (fallocBlock*)((__int8*)block + sizeof(fallocBlock) + blockSize);
		block->magic = FALLOC_MAGIC;
		block->count = 1;
		block->reserved = nullptr;
	}
	{ // critical
		block->next = heap->freeBlocks;
		heap->freeBlocks = block;
	}
}


//////////////////////
// ALLOC

__device__ static fallocCtx* fallocCreateCtx(fallocHeap* heap)
{
	size_t blockSize = heap->blockSize;
	if (sizeof(fallocCtx) > blockSize)
		__THROW;
	fallocCtx* ctx = (fallocCtx*)fallocGetBlock(heap);
	if (!ctx)
		return nullptr;
	ctx->heap = heap;
	unsigned short freeOffset = ctx->node.freeOffset = sizeof(fallocCtx);
	ctx->node.magic = FALLOCNODE_MAGIC;
	ctx->node.next = nullptr; ctx->nodes = (fallocNode*)ctx;
	ctx->node.nextAvailable = nullptr; ctx->availableNodes = (fallocNode*)ctx;
	// close node
	if ((freeOffset + FALLOCNODE_SLACK) > blockSize)
		ctx->availableNodes = nullptr;
	return ctx;
}

__device__ static void fallocDisposeCtx(fallocCtx* ctx)
{
	fallocHeap* heap = ctx->heap;
	for (fallocNode* node = ctx->nodes; node; node = node->next)
		fallocFreeBlock(heap, node);
}

__device__ static void* falloc(fallocCtx* ctx, unsigned short bytes, bool alloc)
{
	if (bytes > (HEAPBLOCK_SIZE - sizeof(fallocCtx)))
		__THROW;
	// find or add available node
	fallocNode* node;
	unsigned short freeOffset;
	unsigned char hasFreeSpace;
	fallocNode* lastNode;
	for (lastNode = (fallocNode*)ctx, node = ctx->availableNodes; node; lastNode = node, node = (alloc ? node->nextAvailable : node->next))
		if (hasFreeSpace = ((freeOffset = (node->freeOffset + bytes)) <= HEAPBLOCK_SIZE))
			break;
	if (!node || !hasFreeSpace) {
		// add node
		node = (fallocNode*)fallocGetBlock(ctx->heap);
		if (!node)
			__THROW;
		freeOffset = node->freeOffset = sizeof(fallocNode); 
		freeOffset += bytes;
		node->magic = FALLOCNODE_MAGIC;
		node->next = ctx->nodes; ctx->nodes = node;
		node->nextAvailable = (alloc ? ctx->availableNodes : nullptr); ctx->availableNodes = node;
	}
	//
	void* obj = (__int8*)node + node->freeOffset;
	node->freeOffset = freeOffset;
	// close node
	if (alloc && ((freeOffset + FALLOCNODE_SLACK) > HEAPBLOCK_SIZE)) {
		if (lastNode == (fallocNode*)ctx)
			ctx->availableNodes = node->nextAvailable;
		else
			lastNode->nextAvailable = node->nextAvailable;
		node->nextAvailable = nullptr;
	}
	return obj;
}

__device__ static void* fallocRetract(fallocCtx* ctx, unsigned short bytes)
{
	fallocNode* node = ctx->availableNodes;
	int freeOffset = (int)node->freeOffset - bytes;
	// multi node, retract node
	if (node != &ctx->node && freeOffset < sizeof(fallocNode)) {
		node->freeOffset = sizeof(fallocNode);
		// search for previous node
		fallocNode* lastNode;
		for (lastNode = (fallocNode*)ctx, node = ctx->nodes; node; lastNode = node, node = node->next)
			if (node == ctx->availableNodes)
				break;
		node = ctx->availableNodes = lastNode;
		freeOffset = (int)node->freeOffset - bytes;
	}
	// first node && !overflow
	if (node == &ctx->node && freeOffset < sizeof(fallocCtx))
		__THROW;
	node->freeOffset = (unsigned short)freeOffset;
	return (__int8*)node + freeOffset;
}

__device__ static void fallocMark(fallocCtx* ctx, void* &mark, unsigned short &mark2) { mark = ctx->availableNodes; mark2 = ctx->availableNodes->freeOffset; }
__device__ static bool fallocAtMark(fallocCtx* ctx, void* mark, unsigned short mark2) { return (mark == ctx->availableNodes && mark2 == ctx->availableNodes->freeOffset); }
