#include "hip/hip_runtime.h"
#include <Core\Core.cu.h>

#pragma region Preamble

#if __HIPCC__
#define TEST(id) \
	__global__ void pcacheTest##id(void *r); \
	void pcacheTest##id##_host(cudaRuntimeHost &r) { pcacheTest##id<<<1, 1>>>(r.heap); cudaRuntimeExecute(r); } \
	__global__ void pcacheTest##id(void *r) \
{ \
	_runtimeSetHeap(r); \
	SysEx::Initialize();
#else
#define TEST(id) \
	__global__ void pcacheTest##id(void *r); \
	void pcacheTest##id##_host(cudaRuntimeHost &r) { pcacheTest##id(r.heap); cudaRuntimeExecute(r); } \
	__global__ void pcacheTest##id(void *r) \
{ \
	_runtimeSetHeap(r); \
	SysEx::Initialize();
#endif

#pragma endregion

//////////////////////////////////////////////////

// printf outputs
TEST(0) {
	_printf("test");
}}