#include "hip/hip_runtime.h"
#include <Core\Core.cu.h>

#pragma region Preamble

#if __HIPCC__
#define TEST(id) \
	__global__ void walTest##id(void *r); \
	void walTest##id##_host(cudaRuntimeHost &r) { walTest##id<<<1, 1>>>(r.heap); cudaRuntimeExecute(r); } \
	__global__ void walTest##id(void *r) \
{ \
	_runtimeSetHeap(r); \
	SysEx::Initialize();
#else
#define TEST(id) \
	__global__ void walTest##id(void *r); \
	void walTest##id##_host(cudaRuntimeHost &r) { walTest##id(r.heap); cudaRuntimeExecute(r); } \
	__global__ void walTest##id(void *r) \
{ \
	_runtimeSetHeap(r); \
	SysEx::Initialize();
#endif

#pragma endregion

//////////////////////////////////////////////////

// printf outputs
TEST(0) {
	_printf("test");
}}
