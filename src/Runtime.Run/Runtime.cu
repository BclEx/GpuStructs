#include "hip/hip_runtime.h"
#include "..\Runtime.src\Runtime.h"
#include "..\Runtime.src\Runtime.cu.h"
__global__ static void runtimeExample(void *r)
{
	_printf("test");
}

void __runtimeExample(cudaRuntimeHost &r)
{
	setRuntimeHeap(r.heap);
	runtimeExample<<<1, 1>>>(r.heap);
	cudaRuntimeExecute(r);
}