#include "hip/hip_runtime.h"
#include <Runtime.cu.h>

__global__ static void runtimeExample0(void *r)
{
	_runtimeSetHeap(r);
	_assert(false);
}

__global__ static void runtimeExample1(void *r)
{
	_runtimeSetHeap(r);
	_printf("t0\n");
	_printf("t1 %s\n", "1");
	_printf("t2 %s %d\n", "1", 2);
	_printf("t3 %s %d %d\n", "1", 2, 3);
	_printf("t4 %s %d %d %d\n", "1", 2, 3, 4);
	_printf("t5 %s %d %d %d %d\n", "1", 2, 3, 4, 5);
	_printf("t6 %s %d %d %d %d %d\n", "1", 2, 3, 4, 5, 6);
	_printf("t7 %s %d %d %d %d %d %d\n", "1", 2, 3, 4, 5, 6, 7);
	_printf("t8 %s %d %d %d %d %d %d %d\n", "1", 2, 3, 4, 5, 6, 7, 8);
	_printf("t9 %s %d %d %d %d %d %d %d %s\n", "1", 2, 3, 4, 5, 6, 7, 8, "9");
	_printf("ta %s %d %d %d %d %d %d %d %d %s\n", "1", 2, 3, 4, 5, 6, 7, 8, 9, "A");
}

__global__ static void runtimeExample2(void *r)
{
	_runtimeSetHeap(r);
	char buf[100];
	__snprintf(buf, sizeof(buf), "t0\n");
	__snprintf(buf, sizeof(buf), "t1 %s\n", "1");
	__snprintf(buf, sizeof(buf), "t2 %s %d\n", "1", 2);
	__snprintf(buf, sizeof(buf), "t3 %s %d %d\n", "1", 2, 3);
	__snprintf(buf, sizeof(buf), "t4 %s %d %d %d\n", "1", 2, 3, 4);
	__snprintf(buf, sizeof(buf), "t5 %s %d %d %d %d\n", "1", 2, 3, 4, 5);
	__snprintf(buf, sizeof(buf), "t6 %s %d %d %d %d %d\n", "1", 2, 3, 4, 5, 6);
	__snprintf(buf, sizeof(buf), "t7 %s %d %d %d %d %d %d\n", "1", 2, 3, 4, 5, 6, 7);
	__snprintf(buf, sizeof(buf), "t8 %s %d %d %d %d %d %d %d\n", "1", 2, 3, 4, 5, 6, 7, 8);
	__snprintf(buf, sizeof(buf), "t9 %s %d %d %d %d %d %d %d %s\n", "1", 2, 3, 4, 5, 6, 7, 8, "9");
	__snprintf(buf, sizeof(buf), "ta %s %d %d %d %d %d %d %d %d %s\n", "1", 2, 3, 4, 5, 6, 7, 8, 9, "A");
}

__global__ static void runtimeExample3(void *r)
{
	_runtimeSetHeap(r);
	_throw("t0\n");
	_throw("t1 %s\n", "1");
	_throw("t2 %s %d\n", "1", 2);
	_throw("t3 %s %d %d\n", "1", 2, 3);
	_throw("t4 %s %d %d %d\n", "1", 2, 3, 4);
}

__global__ static void runtimeExample4(void *r)
{
	_runtimeSetHeap(r);
	char a0 = __toupper('a'); char a0n = __toupper('A'); _assert(a0 == 'A' || a0n == 'A');
	bool a1 = _isspace('a'); bool a1n = _isspace(' '); _assert(!a1 && a1n);
	bool a2 = _isalnum('a'); bool a2n = _isalnum('1'); _assert(a2 && a2n);
	bool a3 = _isalpha('a'); bool a3n = _isalpha('A'); _assert(a3 && a3n);
	bool a4 = _isdigit('a'); bool a4n = _isdigit('1'); _assert(!a4 && a4n);
	bool a5 = _isxdigit('a'); bool a5n = _isxdigit('A'); _assert(!a5 && !a5n);
	char a6 = __tolower('a'); char a6n = __tolower('A'); _assert(a6 == 'a' && a6n == 'a');
}

__global__ static void runtimeExample5(void *r)
{
	_runtimeSetHeap(r);
	array_t<char> name = "SkyM";
	name = "ScottP";
	char *a0 = name;
	size_t a0l = name.length;
}

__global__ static void runtimeExample6(void *r)
{
	_runtimeSetHeap(r);
	char buf[100];
	int a0 = _strcmp("Test", "Test"); _assert(!a0);
	int a1 = _strncmp("Tesa", "Tesb", 3); _assert(!a1);
	_memcpy(buf, "Test", 4);
	_memset(buf, 0, sizeof(buf));
	int a2 = _memcmp("Test", "Test", 4); _assert(!a2);
	int a3 = _strlen30("Test"); _assert(a3 == 4);
}

void __runtimeExample(cudaRuntimeHost &r)
{
	cudaRuntimeSetHeap(r.heap);
	runtimeExample0<<<1, 1>>>(r.heap); cudaRuntimeExecute(r);
	runtimeExample1<<<1, 1>>>(r.heap); cudaRuntimeExecute(r);
	runtimeExample2<<<1, 1>>>(r.heap); cudaRuntimeExecute(r);
	runtimeExample3<<<1, 1>>>(r.heap); cudaRuntimeExecute(r);
	runtimeExample4<<<1, 1>>>(r.heap); cudaRuntimeExecute(r);
	runtimeExample5<<<1, 1>>>(r.heap); cudaRuntimeExecute(r);
	runtimeExample6<<<1, 1>>>(r.heap); cudaRuntimeExecute(r);
}