﻿// status.c
#include "Core.cu.h"

namespace Core
{
	typedef struct StatType StatType;
	__device__ static struct StatType
	{
		int nowValue[10];         // Current value
		int mxValue[10];          // Maximum value
	} Stat = { {0,}, {0,} };

	__device__ int StatusEx::StatusValue(StatusEx::STATUS op)
	{
		_assert(op >= 0 && op < __arrayStaticLength(Stat.nowValue));
		return Stat.nowValue[op];
	}

	__device__ void StatusEx::StatusAdd(StatusEx::STATUS op, int N)
	{
		_assert(op >= 0 && op < __arrayStaticLength(Stat.nowValue));
		Stat.nowValue[op] += N;
		if (Stat.nowValue[op] > Stat.mxValue[op])
			Stat.mxValue[op] = Stat.nowValue[op];
	}

	__device__ void StatusEx::StatusSet(StatusEx::STATUS op, int X)
	{
		_assert(op >= 0 && op < __arrayStaticLength(Stat.nowValue));
		Stat.nowValue[op] = X;
		if (Stat.nowValue[op] > Stat.mxValue[op])
			Stat.mxValue[op] = Stat.nowValue[op];
	}

	__device__ int StatusEx::Status(StatusEx::STATUS op, int *current, int *highwater, int resetFlag)
	{
		if (op < 0 || op >= __arrayStaticLength(Stat.nowValue))
			return SysEx_MISUSE_BKPT;
		*current = Stat.nowValue[op];
		*highwater = Stat.mxValue[op];
		if (resetFlag)
			Stat.mxValue[op] = Stat.nowValue[op];
		return RC::OK;
	}
}
